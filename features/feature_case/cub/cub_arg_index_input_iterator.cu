#include "hip/hip_runtime.h"
#include <hipcub/hipcub.hpp>
#include <hip/hip_runtime.h>
#include <iostream>
#include <limits>
#include <numeric>
#include <random>
#include <type_traits>
#include <vector>

inline std::vector<int> generate_random(size_t N, int Low, int High) {
  std::vector<int> Vec(N, 0);
  std::random_device Dev;
  std::mt19937 Rng(Dev());
  std::uniform_int_distribution<> Dist;
  for (size_t I = 0; I < N; ++I)
    Vec[I] = Dist(Rng);
  return Vec;
}

template <typename T> inline T *safe_device_malloc(size_t Num = 1) {
  T *Ptr = nullptr;
  hipError_t Err = hipMalloc<T>(&Ptr, sizeof(T) * Num);
  if (Err != hipSuccess) {
    std::cerr << "hipMalloc failed: " << hipGetErrorString(Err) << std::endl;
    abort();
  }
  return Ptr;
}

inline void safe_host_copy_to_device(void *Dst, void *Src, size_t Size) {
  hipError_t Err = hipMemcpy(Dst, Src, Size, hipMemcpyHostToDevice);
  if (Err != hipSuccess) {
    std::cerr << "hipMemcpy failed: " << hipGetErrorString(Err) << std::endl;
    abort();
  }
}

inline void safe_device_copy_to_host(void *Dst, void *Src, size_t Size) {
  hipError_t Err = hipMemcpy(Dst, Src, Size, hipMemcpyDeviceToHost);
  if (Err != hipSuccess) {
    std::cerr << "hipMemcpy failed: " << hipGetErrorString(Err) << std::endl;
    abort();
  }
}

inline int *generate_device_random(size_t N, int Low, int High) {
  std::vector<int> Vec = generate_random<int>(N, Low, High);
  int *Buffer = safe_device_malloc<int>(N);
  safe_host_copy_to_device(Buffer, Vec.data(), N * sizeof(int));
  return Buffer;
}

void host() {
  size_t N = 1000;
  std::vector<int> Input = generate_random<int>(N, 1, 100000);
  hipcub::ArgIndexInputIterator<int *> Iter(Input.data());
  for (size_t I = 0; I < N; ++I, ++Iter) {
    const auto &P = *Iter;
    if (Input[P.key] != P.value)
      abort();
  }
}

__global__ void device_kernel(int *Input, bool *Ret, size_t N) {
  hipcub::ArgIndexInputIterator<int *> Iter(Input);
  for (size_t I = 0; I < N; ++I, ++Iter) {
    const auto &P = *Iter;
    if (Input[P.key] != P.value) {
      *Ret = false;
      return;
    }
  }
  *Ret = true;
}

void device() {
  bool HostRet;
  size_t N = 1000;
  int *Buffer = generate_device_random(N, 1, 10000);
  bool *Ret = safe_device_malloc<bool>(1);
  device_kernel<<<1, 1>>>(Buffer, Ret, N);
  safe_device_copy_to_host(&HostRet, Ret, 1);
  if (!HostRet)
    abort();
}

__global__ void host_to_device_kernel(hipcub::ArgIndexInputIterator<int *> Iter,
                                      int *Input, bool *Ret, size_t N) {
  for (size_t I = 0; I < N; ++I, ++Iter) {
    const auto &P = *Iter;
    if (Input[P.key] != P.value) {
      *Ret = false;
      return;
    }
  }
  *Ret = true;
}

void host_to_device() {
  bool HostRet;
  size_t N = 1000;
  int *Buffer = generate_device_random(N, 1, 10000);
  bool *Ret = safe_device_malloc<bool>(1);
  hipcub::ArgIndexInputIterator<int *> Iter(Buffer);
  host_to_device_kernel<<<1, 1>>>(Iter, Buffer, Ret, N);
  safe_device_copy_to_host(&HostRet, Ret, 1);
  if (!HostRet)
    abort();
}

int main() {
  host();
  device();
  host_to_device();
  std::cout << "hipcub::ArgIndexInputIterator pass\n";
  return 0;
}
