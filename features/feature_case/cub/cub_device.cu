// ====------ cub_device.cu------------------------------ *- CUDA -* ------===//
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
//
// ===----------------------------------------------------------------------===//

#include <iostream>
#include <vector>

#include <hip/hip_runtime.h>
#include <hipcub/hipcub.hpp>

#define DATA_NUM 100

template<typename T = int>
void init_data(T* data, int num) {
  T host_data[DATA_NUM];
  for(int i = 0; i < num; i++)
    host_data[i] = i;
  hipMemcpy(data, host_data, num * sizeof(T), hipMemcpyHostToDevice);
}
template<typename T = int>
bool verify_data(T* data, T* expect, int num, int step = 1) {
  T host_data[DATA_NUM];
  hipMemcpy(host_data, data, num * sizeof(T), hipMemcpyDeviceToHost);
  for(int i = 0; i < num; i = i + step) {
    if(host_data[i] != expect[i]) {
      return false;
    }
  }
  return true;
}
template<typename T = int>
void print_data(T* data, int num, bool IsHost = false) {
  if(IsHost) {
    for (int i = 0; i < num; i++) {
      std::cout << data[i] << ", ";
      if((i+1)%32 == 0)
        std::cout << std::endl;
    }
    std::cout << std::endl;
    return;
  }
  T host_data[DATA_NUM];
  hipMemcpy(host_data, data, num * sizeof(int), hipMemcpyDeviceToHost);
  for (int i = 0; i < num; i++) {
    std::cout << host_data[i] << ", ";
    if((i+1)%32 == 0)
        std::cout << std::endl;
  }
  std::cout << std::endl;
}

bool test_reduce_1(){
  int          num_segments = 10;
  int          *device_offsets;
  int          *device_in;
  int          *device_out;
  int          initial_value = INT_MAX;
  void     *temp_storage = NULL;
  size_t   temp_storage_size = 0;
  int expect[DATA_NUM] = {0, 10, 20, 30, 40, 50, 60, 70, 80, 90};

  hipMalloc(&device_offsets, (num_segments + 1) * sizeof(int));
  hipMalloc(&device_in, DATA_NUM * sizeof(int));
  hipMalloc(&device_out, num_segments * sizeof(int));
  init_data(device_in, DATA_NUM);
  int host_offsets[10];
  for(int i = 0; i < num_segments + 1; i++) {
    host_offsets[i] = i * 10;
  }
  hipMemcpy(device_offsets, host_offsets, 11 * sizeof(int), hipMemcpyHostToDevice);

  hipcub::DeviceSegmentedReduce::Reduce(temp_storage, temp_storage_size, device_in, device_out, num_segments, device_offsets, device_offsets + 1, hipcub::Min(), initial_value);

  hipMalloc(&temp_storage, temp_storage_size);

  hipcub::DeviceSegmentedReduce::Reduce(temp_storage, temp_storage_size, device_in, device_out, num_segments, device_offsets, device_offsets + 1, hipcub::Min(), initial_value);

  hipDeviceSynchronize();

  if(!verify_data(device_out, expect, num_segments)) {
    std::cout << "Reduce" << " verify failed" << std::endl;
    std::cout << "expect:" << std::endl;
    print_data<int>(expect, num_segments, true);
    std::cout << "current result:" << std::endl;
    print_data<int>(device_out, num_segments);
    return false;
  }
  return true;
}


bool test_sum_1(){
  int          num_segments = 10;
  int          *device_offsets;
  int          *device_in;
  int          *device_out;
  void     *temp_storage = NULL;
  size_t   temp_storage_size = 0;
  int expect[DATA_NUM] = {45, 145, 245, 345, 445, 545, 645, 745, 845, 945};

  hipMalloc(&device_offsets, (num_segments + 1) * sizeof(int));
  hipMalloc(&device_in, DATA_NUM * sizeof(int));
  hipMalloc(&device_out, num_segments * sizeof(int));
  init_data(device_in, DATA_NUM);
  int host_offsets[10];
  for(int i = 0; i < num_segments + 1; i++) {
    host_offsets[i] = i * 10;
  }
  hipMemcpy(device_offsets, host_offsets, 11 * sizeof(int), hipMemcpyHostToDevice);

  hipcub::DeviceSegmentedReduce::Sum(temp_storage, temp_storage_size, device_in, device_out, num_segments, device_offsets, device_offsets + 1);

  hipMalloc(&temp_storage, temp_storage_size);

  hipcub::DeviceSegmentedReduce::Sum(temp_storage, temp_storage_size, device_in, device_out, num_segments, device_offsets, device_offsets + 1);

  hipDeviceSynchronize();

  if(!verify_data(device_out, expect, num_segments)) {
    std::cout << "Sum" << " verify failed" << std::endl;
    std::cout << "expect:" << std::endl;
    print_data<int>(expect, num_segments, true);
    std::cout << "current result:" << std::endl;
    print_data<int>(device_out, num_segments);
    return false;
  }
  return true;
}

bool test_sum_2(){
  int          num_segments = 10;
  int          *device_offsets;
  int          *device_in;
  int          *device_out;
  void     *temp_storage = NULL;
  size_t   temp_storage_size = 0;
  int expect[DATA_NUM] = {190, 0, 245, 345, 445, 545, 645, 745, 845, 945};

  hipMalloc(&device_offsets, (num_segments + 1) * sizeof(int));
  hipMalloc(&device_in, DATA_NUM * sizeof(int));
  hipMalloc(&device_out, num_segments * sizeof(int));
  init_data(device_in, DATA_NUM);
  int host_offsets[10];
  for(int i = 0; i < num_segments + 1; i++) {
    host_offsets[i] = i * 10;
  }
  host_offsets[1] = 20;
  hipMemcpy(device_offsets, host_offsets, 11 * sizeof(int), hipMemcpyHostToDevice);
  hipcub::DeviceSegmentedReduce::Sum(temp_storage, temp_storage_size, device_in, device_out, num_segments, device_offsets, device_offsets + 1);

  hipMalloc(&temp_storage, temp_storage_size);

  hipcub::DeviceSegmentedReduce::Sum(temp_storage, temp_storage_size, device_in, device_out, num_segments, device_offsets, device_offsets + 1);

  hipDeviceSynchronize();

  if(!verify_data(device_out, expect, num_segments)) {
    std::cout << "Sum" << " verify failed" << std::endl;
    std::cout << "expect:" << std::endl;
    print_data<int>(expect, num_segments, true);
    std::cout << "current result:" << std::endl;
    print_data<int>(device_out, num_segments);
    return false;
  }
  return true;
}

bool test_min(){
  int          num_segments = 10;
  int          *device_offsets;
  int          *device_in;
  int          *device_out;
  void     *temp_storage = NULL;
  size_t   temp_storage_size = 0;
  int expect[DATA_NUM] = {0, 10, 20, 30, 40, 50, 60, 70, 80, 90};

  hipMalloc(&device_offsets, (num_segments + 1) * sizeof(int));
  hipMalloc(&device_in, DATA_NUM * sizeof(int));
  hipMalloc(&device_out, num_segments * sizeof(int));
  init_data(device_in, DATA_NUM);
  int host_offsets[10];
  for(int i = 0; i < num_segments + 1; i++) {
    host_offsets[i] = i * 10;
  }
  hipMemcpy(device_offsets, host_offsets, 11 * sizeof(int), hipMemcpyHostToDevice);

  hipcub::DeviceSegmentedReduce::Min(temp_storage, temp_storage_size, device_in, device_out, num_segments, device_offsets, device_offsets + 1);

  hipMalloc(&temp_storage, temp_storage_size);

  hipcub::DeviceSegmentedReduce::Min(temp_storage, temp_storage_size, device_in, device_out, num_segments, device_offsets, device_offsets + 1);

  hipDeviceSynchronize();

  if(!verify_data(device_out, expect, num_segments)) {
    std::cout << "Min" << " verify failed" << std::endl;
    std::cout << "expect:" << std::endl;
    print_data<int>(expect, num_segments, true);
    std::cout << "current result:" << std::endl;
    print_data<int>(device_out, num_segments);
    return false;
  }
  return true;
}


bool test_max(){
  int          num_segments = 10;
  int          *device_offsets;
  int          *device_in;
  int          *device_out;
  void     *temp_storage = NULL;
  size_t   temp_storage_size = 0;
  int expect[DATA_NUM] = {9, 19, 29, 39, 49, 59, 69, 79, 89, 99};

  hipMalloc(&device_offsets, (num_segments + 1) * sizeof(int));
  hipMalloc(&device_in, DATA_NUM * sizeof(int));
  hipMalloc(&device_out, num_segments * sizeof(int));
  init_data(device_in, DATA_NUM);

  int host_offsets[10];
  for(int i = 0; i < num_segments + 1; i++) {
    host_offsets[i] = i * 10;
  }
  hipMemcpy(device_offsets, host_offsets, 11 * sizeof(int), hipMemcpyHostToDevice);

  hipcub::DeviceSegmentedReduce::Max(temp_storage, temp_storage_size, device_in, device_out, num_segments, device_offsets, device_offsets + 1);

  hipMalloc(&temp_storage, temp_storage_size);

  hipcub::DeviceSegmentedReduce::Max(temp_storage, temp_storage_size, device_in, device_out, num_segments, device_offsets, device_offsets + 1);

  hipDeviceSynchronize();

  if(!verify_data(device_out, expect, num_segments)) {
    std::cout << "Max" << " verify failed" << std::endl;
    std::cout << "expect:" << std::endl;
    print_data<int>(expect, num_segments, true);
    std::cout << "current result:" << std::endl;
    print_data<int>(device_out, num_segments);
    return false;
  }
  return true;
}

/// hipcub::DeviceReduce::Sum
bool test_device_reduce_sum() {
  int *device_in;
  int *device_out;
  void *temp_storage = NULL;
  size_t temp_storage_size = 0;
  int expect = 4950;
  hipMalloc((void **)&device_in, sizeof(int) * DATA_NUM);
  hipMalloc((void **)&device_out, sizeof(int));
  init_data(device_in, DATA_NUM);
  hipcub::DeviceReduce::Sum(temp_storage, temp_storage_size, device_in,
                         device_out, DATA_NUM);
  hipMalloc((void **)&temp_storage, temp_storage_size);
  hipcub::DeviceReduce::Sum(temp_storage, temp_storage_size, device_in,
                         device_out, DATA_NUM);
  hipDeviceSynchronize();
  if (!verify_data(device_out, &expect, 1)) {
    std::cout << "hipcub::DeviceReduce::Sum verify failed\n";
    std::cout << "expect:\n";
    print_data<int>(&expect, 1, true);
    std::cout << "current result:\n";
    print_data<int>(device_out, 1);
    return false;
  }
  return true;
}

// hipcub::DeviceScan::InclusiveSum
bool test_device_scan_inclusive_sum() {
  static const int n = 10;
  int *device_in;
  int *device_out;
  void *temp_storage = NULL;
  size_t temp_storage_size = 0;
  int expect[n] = {0, 1, 3, 6, 10, 15, 21, 28, 36, 45};
  hipMalloc((void **)&device_in, sizeof(int) * n);
  hipMalloc((void **)&device_out, sizeof(int) * n);
  init_data(device_in, n);
  hipcub::DeviceScan::InclusiveSum(temp_storage, temp_storage_size, device_in,
                                device_out, n);
  hipMalloc((void **)&temp_storage, temp_storage_size);
  hipcub::DeviceScan::InclusiveSum(temp_storage, temp_storage_size, device_in,
                                device_out, n);
  hipDeviceSynchronize();
  if (!verify_data(device_out, expect, n)) {
    std::cout << "hipcub::DeviceScan::InclusiveSum verify failed\n";
    std::cout << "expect:\n";
    print_data<int>(expect, 1, true);
    std::cout << "current result:\n";
    print_data<int>(device_out, n);
    return false;
  }
  return true;
}

// hipcub::DeviceScan::ExclusiveSum
bool test_device_scan_exclusive_sum() {
  static const int n = 10;
  int *device_in;
  int *device_out;
  void *temp_storage = NULL;
  size_t temp_storage_size = 0;
  int expect[n] = {0, 0, 1, 3, 6, 10, 15, 21, 28, 36};
  hipMalloc((void **)&device_in, sizeof(int) * n);
  hipMalloc((void **)&device_out, sizeof(int) * n);
  init_data(device_in, n);
  hipcub::DeviceScan::ExclusiveSum(temp_storage, temp_storage_size, device_in,
                                device_out, n);
  hipMalloc((void **)&temp_storage, temp_storage_size);
  hipcub::DeviceScan::ExclusiveSum(temp_storage, temp_storage_size, device_in,
                                device_out, n);
  hipDeviceSynchronize();
  if (!verify_data(device_out, expect, n)) {
    std::cout << "hipcub::DeviceScan::InclusiveSum verify failed\n";
    std::cout << "expect:\n";
    print_data<int>(expect, 1, true);
    std::cout << "current result:\n";
    print_data<int>(device_out, n);
    return false;
  }
  return true;
}

// hipcub::DeviceSelect::Flagged
bool test_device_select_flagged() {
  static const int n = 10;
  int *device_in = nullptr;
  int *device_out = nullptr;
  int *device_flagged = nullptr;
  int *device_select_num = nullptr;
  int *device_tmp = nullptr;
  size_t n_device_tmp = 0;
  int host_in[n] = {0, 1, 2, 3, 4, 5, 6, 7, 8, 9};
  int host_flagged[n] = {0, 1, 0, 1, 0, 1, 0, 1, 0, 1};
  int expect_out[] = {1, 3, 5, 7, 9};
  int expect_select_num = 5;
  hipMalloc((void **)&device_in, n * sizeof(int));
  hipMalloc((void **)&device_out, n * sizeof(int));
  hipMalloc((void **)&device_flagged, n * sizeof(int));
  hipMalloc((void **)&device_select_num, sizeof(int));
  hipMemcpy(device_in, (void *)host_in, sizeof(host_in),
             hipMemcpyHostToDevice);
  hipMemcpy(device_flagged, host_flagged, sizeof(host_flagged),
             hipMemcpyHostToDevice);
  hipcub::DeviceSelect::Flagged(device_tmp, n_device_tmp, device_in,
                             device_flagged, device_out, device_select_num,
                             n);
  hipMalloc((void **)&device_tmp, n_device_tmp);
  hipcub::DeviceSelect::Flagged(device_tmp, n_device_tmp, device_in,
                             device_flagged, device_out, device_select_num,
                             n);
  hipDeviceSynchronize();

  if (!verify_data(device_select_num, &expect_select_num, 1)) {
    std::cout << "hipcub::DeviceScan::InclusiveSum select_num verify failed\n";
    std::cout << "expect:\n";
    print_data<int>(&expect_select_num, 1, true);
    std::cout << "current result:\n";
    print_data<int>(device_select_num, 1);
    return false;
  }

  if (!verify_data(device_out, (int *)expect_out, expect_select_num)) {
    std::cout << "hipcub::DeviceScan::InclusiveSum output data verify failed\n";
    std::cout << "expect:\n";
    print_data<int>(expect_out, 1, true);
    std::cout << "current result:\n";
    print_data<int>(device_out, 1);
    return false;
  }
  return true;
}

int main() {
  bool Result = true;
  Result = test_reduce_1() && Result;
  Result = test_sum_1() && Result;
  Result = test_sum_2() && Result;
  Result = test_min() && Result;
  Result = test_max() && Result;
  Result = test_device_reduce_sum() && Result;
  Result = test_device_scan_inclusive_sum() && Result;
  Result = test_device_scan_inclusive_sum() && Result;
  Result = test_device_select_flagged() && Result;
  if(Result) {
    std::cout << "cub_device Pass" << std::endl;
  }
  return 0;
}

