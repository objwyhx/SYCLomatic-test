// ====------ cub_device.cu------------------------------ *- CUDA -* ------===//
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
//
// ===----------------------------------------------------------------------===//

#include <iostream>
#include <vector>

#include <hip/hip_runtime.h>
#include <hipcub/hipcub.hpp>

#define DATA_NUM 100

struct ScanOp {
  template <typename T, typename = typename std::enable_if<
                            std::is_arithmetic<T>::value>::type>
  __device__ T operator()(const T &lhs, const T &rhs) const {
    return lhs + rhs;
  }
};

template<typename T = int>
void init_data(T* data, int num) {
  T host_data[DATA_NUM];
  for(int i = 0; i < num; i++)
    host_data[i] = i;
  hipMemcpy(data, host_data, num * sizeof(T), hipMemcpyHostToDevice);
}
template<typename T = int>
bool verify_data(T* data, T* expect, int num, int step = 1) {
  T host_data[DATA_NUM];
  hipMemcpy(host_data, data, num * sizeof(T), hipMemcpyDeviceToHost);
  for(int i = 0; i < num; i = i + step) {
    if(host_data[i] != expect[i]) {
      return false;
    }
  }
  return true;
}
template<typename T = int>
void print_data(T* data, int num, bool IsHost = false) {
  if(IsHost) {
    for (int i = 0; i < num; i++) {
      std::cout << data[i] << ", ";
      if((i+1)%32 == 0)
        std::cout << std::endl;
    }
    std::cout << std::endl;
    return;
  }
  T host_data[DATA_NUM];
  hipMemcpy(host_data, data, num * sizeof(int), hipMemcpyDeviceToHost);
  for (int i = 0; i < num; i++) {
    std::cout << host_data[i] << ", ";
    if((i+1)%32 == 0)
        std::cout << std::endl;
  }
  std::cout << std::endl;
}

bool test_reduce_1(){
  int          num_segments = 10;
  int          *device_offsets;
  int          *device_in;
  int          *device_out;
  int          initial_value = INT_MAX;
  void     *temp_storage = NULL;
  size_t   temp_storage_size = 0;
  int expect[DATA_NUM] = {0, 10, 20, 30, 40, 50, 60, 70, 80, 90};

  hipMalloc(&device_offsets, (num_segments + 1) * sizeof(int));
  hipMalloc(&device_in, DATA_NUM * sizeof(int));
  hipMalloc(&device_out, num_segments * sizeof(int));
  init_data(device_in, DATA_NUM);
  int host_offsets[10];
  for(int i = 0; i < num_segments + 1; i++) {
    host_offsets[i] = i * 10;
  }
  hipMemcpy(device_offsets, host_offsets, 11 * sizeof(int), hipMemcpyHostToDevice);

  hipcub::DeviceSegmentedReduce::Reduce(temp_storage, temp_storage_size, device_in, device_out, num_segments, device_offsets, device_offsets + 1, hipcub::Min(), initial_value);

  hipMalloc(&temp_storage, temp_storage_size);

  hipcub::DeviceSegmentedReduce::Reduce(temp_storage, temp_storage_size, device_in, device_out, num_segments, device_offsets, device_offsets + 1, hipcub::Min(), initial_value);

  hipDeviceSynchronize();

  if(!verify_data(device_out, expect, num_segments)) {
    std::cout << "Reduce" << " verify failed" << std::endl;
    std::cout << "expect:" << std::endl;
    print_data<int>(expect, num_segments, true);
    std::cout << "current result:" << std::endl;
    print_data<int>(device_out, num_segments);
    return false;
  }
  return true;
}


bool test_sum_1(){
  int          num_segments = 10;
  int          *device_offsets;
  int          *device_in;
  int          *device_out;
  void     *temp_storage = NULL;
  size_t   temp_storage_size = 0;
  int expect[DATA_NUM] = {45, 145, 245, 345, 445, 545, 645, 745, 845, 945};

  hipMalloc(&device_offsets, (num_segments + 1) * sizeof(int));
  hipMalloc(&device_in, DATA_NUM * sizeof(int));
  hipMalloc(&device_out, num_segments * sizeof(int));
  init_data(device_in, DATA_NUM);
  int host_offsets[10];
  for(int i = 0; i < num_segments + 1; i++) {
    host_offsets[i] = i * 10;
  }
  hipMemcpy(device_offsets, host_offsets, 11 * sizeof(int), hipMemcpyHostToDevice);

  hipcub::DeviceSegmentedReduce::Sum(temp_storage, temp_storage_size, device_in, device_out, num_segments, device_offsets, device_offsets + 1);

  hipMalloc(&temp_storage, temp_storage_size);

  hipcub::DeviceSegmentedReduce::Sum(temp_storage, temp_storage_size, device_in, device_out, num_segments, device_offsets, device_offsets + 1);

  hipDeviceSynchronize();

  if(!verify_data(device_out, expect, num_segments)) {
    std::cout << "Sum" << " verify failed" << std::endl;
    std::cout << "expect:" << std::endl;
    print_data<int>(expect, num_segments, true);
    std::cout << "current result:" << std::endl;
    print_data<int>(device_out, num_segments);
    return false;
  }
  return true;
}

bool test_sum_2(){
  int          num_segments = 10;
  int          *device_offsets;
  int          *device_in;
  int          *device_out;
  void     *temp_storage = NULL;
  size_t   temp_storage_size = 0;
  int expect[DATA_NUM] = {190, 0, 245, 345, 445, 545, 645, 745, 845, 945};

  hipMalloc(&device_offsets, (num_segments + 1) * sizeof(int));
  hipMalloc(&device_in, DATA_NUM * sizeof(int));
  hipMalloc(&device_out, num_segments * sizeof(int));
  init_data(device_in, DATA_NUM);
  int host_offsets[10];
  for(int i = 0; i < num_segments + 1; i++) {
    host_offsets[i] = i * 10;
  }
  host_offsets[1] = 20;
  hipMemcpy(device_offsets, host_offsets, 11 * sizeof(int), hipMemcpyHostToDevice);
  hipcub::DeviceSegmentedReduce::Sum(temp_storage, temp_storage_size, device_in, device_out, num_segments, device_offsets, device_offsets + 1);

  hipMalloc(&temp_storage, temp_storage_size);

  hipcub::DeviceSegmentedReduce::Sum(temp_storage, temp_storage_size, device_in, device_out, num_segments, device_offsets, device_offsets + 1);

  hipDeviceSynchronize();

  if(!verify_data(device_out, expect, num_segments)) {
    std::cout << "Sum" << " verify failed" << std::endl;
    std::cout << "expect:" << std::endl;
    print_data<int>(expect, num_segments, true);
    std::cout << "current result:" << std::endl;
    print_data<int>(device_out, num_segments);
    return false;
  }
  return true;
}

bool test_min(){
  int          num_segments = 10;
  int          *device_offsets;
  int          *device_in;
  int          *device_out;
  void     *temp_storage = NULL;
  size_t   temp_storage_size = 0;
  int expect[DATA_NUM] = {0, 10, 20, 30, 40, 50, 60, 70, 80, 90};

  hipMalloc(&device_offsets, (num_segments + 1) * sizeof(int));
  hipMalloc(&device_in, DATA_NUM * sizeof(int));
  hipMalloc(&device_out, num_segments * sizeof(int));
  init_data(device_in, DATA_NUM);
  int host_offsets[10];
  for(int i = 0; i < num_segments + 1; i++) {
    host_offsets[i] = i * 10;
  }
  hipMemcpy(device_offsets, host_offsets, 11 * sizeof(int), hipMemcpyHostToDevice);

  hipcub::DeviceSegmentedReduce::Min(temp_storage, temp_storage_size, device_in, device_out, num_segments, device_offsets, device_offsets + 1);

  hipMalloc(&temp_storage, temp_storage_size);

  hipcub::DeviceSegmentedReduce::Min(temp_storage, temp_storage_size, device_in, device_out, num_segments, device_offsets, device_offsets + 1);

  hipDeviceSynchronize();

  if(!verify_data(device_out, expect, num_segments)) {
    std::cout << "Min" << " verify failed" << std::endl;
    std::cout << "expect:" << std::endl;
    print_data<int>(expect, num_segments, true);
    std::cout << "current result:" << std::endl;
    print_data<int>(device_out, num_segments);
    return false;
  }
  return true;
}


bool test_max(){
  int          num_segments = 10;
  int          *device_offsets;
  int          *device_in;
  int          *device_out;
  void     *temp_storage = NULL;
  size_t   temp_storage_size = 0;
  int expect[DATA_NUM] = {9, 19, 29, 39, 49, 59, 69, 79, 89, 99};

  hipMalloc(&device_offsets, (num_segments + 1) * sizeof(int));
  hipMalloc(&device_in, DATA_NUM * sizeof(int));
  hipMalloc(&device_out, num_segments * sizeof(int));
  init_data(device_in, DATA_NUM);

  int host_offsets[10];
  for(int i = 0; i < num_segments + 1; i++) {
    host_offsets[i] = i * 10;
  }
  hipMemcpy(device_offsets, host_offsets, 11 * sizeof(int), hipMemcpyHostToDevice);

  hipcub::DeviceSegmentedReduce::Max(temp_storage, temp_storage_size, device_in, device_out, num_segments, device_offsets, device_offsets + 1);

  hipMalloc(&temp_storage, temp_storage_size);

  hipcub::DeviceSegmentedReduce::Max(temp_storage, temp_storage_size, device_in, device_out, num_segments, device_offsets, device_offsets + 1);

  hipDeviceSynchronize();

  if(!verify_data(device_out, expect, num_segments)) {
    std::cout << "Max" << " verify failed" << std::endl;
    std::cout << "expect:" << std::endl;
    print_data<int>(expect, num_segments, true);
    std::cout << "current result:" << std::endl;
    print_data<int>(device_out, num_segments);
    return false;
  }
  return true;
}

/// hipcub::DeviceReduce::Sum
bool test_device_reduce_sum() {
  int *device_in;
  int *device_out;
  void *temp_storage = NULL;
  size_t temp_storage_size = 0;
  int expect = 4950;
  hipMalloc((void **)&device_in, sizeof(int) * DATA_NUM);
  hipMalloc((void **)&device_out, sizeof(int));
  init_data(device_in, DATA_NUM);
  hipcub::DeviceReduce::Sum(temp_storage, temp_storage_size, device_in,
                         device_out, DATA_NUM);
  hipMalloc((void **)&temp_storage, temp_storage_size);
  hipcub::DeviceReduce::Sum(temp_storage, temp_storage_size, device_in,
                         device_out, DATA_NUM);
  hipDeviceSynchronize();
  if (!verify_data(device_out, &expect, 1)) {
    std::cout << "hipcub::DeviceReduce::Sum verify failed\n";
    std::cout << "expect:\n";
    print_data<int>(&expect, 1, true);
    std::cout << "current result:\n";
    print_data<int>(device_out, 1);
    return false;
  }
  return true;
}

// hipcub::DeviceScan::InclusiveSum
bool test_device_scan_inclusive_sum() {
  static const int n = 10;
  int *device_in;
  int *device_out;
  void *temp_storage = NULL;
  size_t temp_storage_size = 0;
  int expect[n] = {0, 1, 3, 6, 10, 15, 21, 28, 36, 45};
  hipMalloc((void **)&device_in, sizeof(int) * n);
  hipMalloc((void **)&device_out, sizeof(int) * n);
  init_data(device_in, n);
  hipcub::DeviceScan::InclusiveSum(temp_storage, temp_storage_size, device_in,
                                device_out, n);
  hipMalloc((void **)&temp_storage, temp_storage_size);
  hipcub::DeviceScan::InclusiveSum(temp_storage, temp_storage_size, device_in,
                                device_out, n);
  hipDeviceSynchronize();
  if (!verify_data(device_out, expect, n)) {
    std::cout << "hipcub::DeviceScan::InclusiveSum verify failed\n";
    std::cout << "expect:\n";
    print_data<int>(expect, 1, true);
    std::cout << "current result:\n";
    print_data<int>(device_out, n);
    return false;
  }
  return true;
}

// hipcub::DeviceScan::ExclusiveSum
bool test_device_scan_exclusive_sum() {
  static const int n = 10;
  int *device_in;
  int *device_out;
  void *temp_storage = NULL;
  size_t temp_storage_size = 0;
  int expect[n] = {0, 0, 1, 3, 6, 10, 15, 21, 28, 36};
  hipMalloc((void **)&device_in, sizeof(int) * n);
  hipMalloc((void **)&device_out, sizeof(int) * n);
  init_data(device_in, n);
  hipcub::DeviceScan::ExclusiveSum(temp_storage, temp_storage_size, device_in,
                                device_out, n);
  hipMalloc((void **)&temp_storage, temp_storage_size);
  hipcub::DeviceScan::ExclusiveSum(temp_storage, temp_storage_size, device_in,
                                device_out, n);
  hipDeviceSynchronize();
  if (!verify_data(device_out, expect, n)) {
    std::cout << "hipcub::DeviceScan::ExclusiveSum verify failed\n";
    std::cout << "expect:\n";
    print_data<int>(expect, 1, true);
    std::cout << "current result:\n";
    print_data<int>(device_out, n);
    return false;
  }
  return true;
}

// hipcub::DeviceSelect::Flagged
bool test_device_select_flagged() {
  static const int n = 10;
  int *device_in = nullptr;
  int *device_out = nullptr;
  int *device_flagged = nullptr;
  int *device_select_num = nullptr;
  int *device_tmp = nullptr;
  size_t n_device_tmp = 0;
  int host_in[n] = {0, 1, 2, 3, 4, 5, 6, 7, 8, 9};
  int host_flagged[n] = {0, 1, 0, 1, 0, 1, 0, 1, 0, 1};
  int expect_out[] = {1, 3, 5, 7, 9};
  int expect_select_num = 5;
  hipMalloc((void **)&device_in, n * sizeof(int));
  hipMalloc((void **)&device_out, n * sizeof(int));
  hipMalloc((void **)&device_flagged, n * sizeof(int));
  hipMalloc((void **)&device_select_num, sizeof(int));
  hipMemcpy(device_in, (void *)host_in, sizeof(host_in),
             hipMemcpyHostToDevice);
  hipMemcpy(device_flagged, host_flagged, sizeof(host_flagged),
             hipMemcpyHostToDevice);
  hipcub::DeviceSelect::Flagged(device_tmp, n_device_tmp, device_in,
                             device_flagged, device_out, device_select_num,
                             n);
  hipMalloc((void **)&device_tmp, n_device_tmp);
  hipcub::DeviceSelect::Flagged(device_tmp, n_device_tmp, device_in,
                             device_flagged, device_out, device_select_num,
                             n);
  hipDeviceSynchronize();

  if (!verify_data(device_select_num, &expect_select_num, 1)) {
    std::cout << "hipcub::DeviceSelect::Flagged select_num verify failed\n";
    std::cout << "expect:\n";
    print_data<int>(&expect_select_num, 1, true);
    std::cout << "current result:\n";
    print_data<int>(device_select_num, 1);
    return false;
  }

  if (!verify_data(device_out, (int *)expect_out, expect_select_num)) {
    std::cout << "hipcub::DeviceSelect::Flagged output data verify failed\n";
    std::cout << "expect:\n";
    print_data<int>(expect_out, 1, true);
    std::cout << "current result:\n";
    print_data<int>(device_out, 1);
    return false;
  }
  return true;
}

// hipcub::DeviceScan::InclusiveScan
bool test_device_inclusive_scan() {
  static const int n = 10;
  int *device_in;
  int *device_out;
  void *temp_storage = NULL;
  size_t temp_storage_size = 0;
  int expect[n] = {0, 1, 3, 6, 10, 15, 21, 28, 36, 45};
  hipMalloc((void **)&device_in, sizeof(int) * n);
  hipMalloc((void **)&device_out, sizeof(int) * n);
  init_data(device_in, n);
  ScanOp scan_op;
  hipcub::DeviceScan::InclusiveScan(temp_storage, temp_storage_size, device_in,
                                device_out, scan_op, n);
  hipMalloc((void **)&temp_storage, temp_storage_size);
  hipcub::DeviceScan::InclusiveScan(temp_storage, temp_storage_size, device_in,
                                device_out, scan_op, n);
  hipDeviceSynchronize();
  if (!verify_data(device_out, expect, n)) {
    std::cout << "hipcub::DeviceScan::InclusiveScan verify failed\n";
    std::cout << "expect:\n";
    print_data<int>(expect, 1, true);
    std::cout << "current result:\n";
    print_data<int>(device_out, n);
    return false;
  }
  return true;
}

// hipcub::DeviceScan::ExclusiveScan
bool test_device_exclusive_scan() {
  static const int n = 10;
  int *device_in;
  int *device_out;
  void *temp_storage = NULL;
  size_t temp_storage_size = 0;
  int expect[n] = {0, 0, 1, 3, 6, 10, 15, 21, 28, 36};
  hipMalloc((void **)&device_in, sizeof(int) * n);
  hipMalloc((void **)&device_out, sizeof(int) * n);
  init_data(device_in, n);
  ScanOp scan_op;
  hipcub::DeviceScan::ExclusiveScan(temp_storage, temp_storage_size, device_in,
                                device_out, scan_op, 0, n);
  hipMalloc((void **)&temp_storage, temp_storage_size);
  hipcub::DeviceScan::ExclusiveScan(temp_storage, temp_storage_size, device_in,
                                device_out, scan_op, 0, n);
  hipDeviceSynchronize();
  if (!verify_data(device_out, expect, n)) {
    std::cout << "hipcub::DeviceScan::ExclusiveScan verify failed\n";
    std::cout << "expect:\n";
    print_data<int>(expect, n, true);
    std::cout << "current result:\n";
    print_data<int>(device_out, n);
    return false;
  }
  return true;
}

// hipcub::DeviceSelect::Unique
bool test_device_unique() {
  static const int N = 8;
  int data[N] = {0, 2, 2, 9, 5, 5, 5, 8};
  int *d_in = nullptr;
  int *d_out = nullptr;
  int *d_temp = nullptr;
  int *d_selected_num = nullptr;
  size_t d_temp_size = 0;
  hipMalloc((void **)&d_in, sizeof(int) * N);
  hipMalloc((void **)&d_out, sizeof(int) * N);
  hipMalloc((void **)&d_selected_num, sizeof(int));
  hipMemcpy((void *)d_in, (void *)data, sizeof(data), hipMemcpyHostToDevice);
  hipcub::DeviceSelect::Unique(nullptr, d_temp_size, d_in, d_out, d_selected_num, N);
  hipMalloc((void **)&d_temp, d_temp_size);
  hipcub::DeviceSelect::Unique((void *)d_temp, d_temp_size, d_in, d_out, d_selected_num, N);
  hipDeviceSynchronize();

  int expect_select_num = 5;
  int expect_out[] = {0, 2, 9, 5, 8};

  if (!verify_data(d_selected_num, &expect_select_num, 1)) {
    std::cout << "hipcub::DeviceSelect::Unique select_num verify failed\n";
    std::cout << "expect:\n";
    print_data<int>(&expect_select_num, 1, true);
    std::cout << "current result:\n";
    print_data<int>(d_selected_num, 1);
    return false;
  }

  if (!verify_data(d_out, (int *)expect_out, expect_select_num)) {
    std::cout << "hipcub::DeviceSelect::Unique output data verify failed\n";
    std::cout << "expect:\n";
    print_data<int>(expect_out, 1, true);
    std::cout << "current result:\n";
    print_data<int>(d_out, 1);
    return false;
  }
  return true;

}

// hipcub::DeviceRunLengthEncode::Encode
bool test_device_encode() {
  static const int N = 8;
  int data[N] = {0, 2, 2, 9, 5, 5, 5, 8};
  int *d_in = nullptr;
  int *d_temp = nullptr;
  int *d_unique = nullptr;
  int *d_counts = nullptr;
  int *d_selected_num = nullptr;
  int h_selected_num = 0;
  size_t d_temp_size = 0;

  hipMalloc((void **)&d_in, sizeof(int) * N);
  hipMalloc((void **)&d_unique, sizeof(int) * N);
  hipMalloc((void **)&d_counts, sizeof(int) * N);
  hipMalloc((void **)&d_selected_num, sizeof(int));
  hipMemcpy((void *)d_in, (void *)data, sizeof(data), hipMemcpyHostToDevice);
  hipcub::DeviceRunLengthEncode::Encode(nullptr, d_temp_size, d_in, d_unique, d_counts, d_selected_num, N);
  hipMalloc((void **)&d_temp, d_temp_size);
  hipcub::DeviceRunLengthEncode::Encode(d_temp, d_temp_size, d_in, d_unique, d_counts, d_selected_num, N);
  hipDeviceSynchronize();

  int expect_select_num = 5;
  int expect_unique[] = {0, 2, 9, 5, 8};
  int expect_counts[] = {1, 2, 1, 3, 1};

  
  if (!verify_data(d_selected_num, &expect_select_num, 1)) {
    std::cout << "hipcub::DeviceRunLengthEncode::Encode select_num verify failed\n";
    std::cout << "expect:\n";
    print_data<int>(&expect_select_num, 1, true);
    std::cout << "current result:\n";
    print_data<int>(d_selected_num, 1);
    return false;
  }

  if (!verify_data(d_unique, (int *)expect_unique, expect_select_num)) {
    std::cout << "hipcub::DeviceRunLengthEncode::Encode output unique data verify failed\n";
    std::cout << "expect:\n";
    print_data<int>(expect_unique, 1, true);
    std::cout << "current result:\n";
    print_data<int>(d_unique, 1);
    return false;
  }

   if (!verify_data(d_counts, (int *)expect_counts, expect_select_num)) {
    std::cout << "hipcub::DeviceRunLengthEncode::Encode output counts data verify failed\n";
    std::cout << "expect:\n";
    print_data<int>(expect_counts, 1, true);
    std::cout << "current result:\n";
    print_data<int>(d_counts, 1);
    return false;
  }

  return true;
}

int main() {
  bool Result = true;
  Result = test_reduce_1() && Result;
  Result = test_sum_1() && Result;
  Result = test_sum_2() && Result;
  Result = test_min() && Result;
  Result = test_max() && Result;
  Result = test_device_reduce_sum() && Result;
  Result = test_device_scan_inclusive_sum() && Result;
  Result = test_device_scan_inclusive_sum() && Result;
  Result = test_device_select_flagged() && Result;
  Result = test_device_exclusive_scan() && Result;
  Result = test_device_inclusive_scan() && Result;
  Result = test_device_unique() && Result;
  Result = test_device_encode() && Result;
  if(Result) {
    std::cout << "cub_device Pass" << std::endl;
  }
  return 0;
}

