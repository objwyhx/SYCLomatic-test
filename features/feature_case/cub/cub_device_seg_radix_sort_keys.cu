// ====------ cub_device_seg_radix_sort_keys.cu ---------- *- CUDA -* ------===//
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
//
// ===----------------------------------------------------------------------===//

#include <cstdlib>
#include <hipcub/hipcub.hpp>
#include <initializer_list>
#include <stdio.h>
#include <vector>

template <typename T> T *init(std::initializer_list<T> list) {
  T *arr = nullptr;
  hipMalloc(&arr, sizeof(T) * list.size());
  hipMemcpy(arr, list.begin(), sizeof(T) * list.size(),
             hipMemcpyHostToDevice);
  return arr;
}

bool test() {
  // clang-format off
  // Declare, allocate, and initialize device-accessible pointers for sorting data
  int  num_items          = 7;
  int  num_segments       = 3;
  int  *d_offsets         = init({0, 3, 3, 7});
  int  *d_keys_in         = init({8, 6, 7, 5, 3, 0, 9});
  int  *d_keys_out        = init({0, 0, 0, 0, 0, 0, 0});
  std::vector<int> expected_keys_out{6, 7, 8, 0, 3, 5, 9};
  // clang-format on

  // Determine temporary device storage requirements
  void *d_temp_storage = NULL;
  size_t temp_storage_bytes = 0;
  hipcub::DeviceSegmentedRadixSort::SortKeys(
      d_temp_storage, temp_storage_bytes, d_keys_in, d_keys_out, num_items,
      num_segments, d_offsets, d_offsets + 1);
  // Allocate temporary storage
  hipMalloc(&d_temp_storage, temp_storage_bytes);
  // Run sorting operation
  hipcub::DeviceSegmentedRadixSort::SortKeys(
      d_temp_storage, temp_storage_bytes, d_keys_in, d_keys_out, num_items,
      num_segments, d_offsets, d_offsets + 1);

  std::vector<int> keys_out(num_items), values_out(num_items);
  hipMemcpy(keys_out.data(), d_keys_out, sizeof(int) * num_items,
             hipMemcpyDeviceToHost);
  hipFree(d_offsets);
  hipFree(d_keys_in);
  hipFree(d_keys_out);
  hipFree(d_temp_storage);
  return std::equal(keys_out.begin(), keys_out.end(),
                    expected_keys_out.begin());
}

bool test1() {
  // clang-format off
  // Declare, allocate, and initialize device-accessible pointers for sorting data
  int  num_items          = 7;
  int  num_segments       = 3;
  int  *d_offsets         = init({0, 3, 3, 7});
  int  *d_keys_in         = init({8, 6, 7, 5, 3, 0, 9});
  int  *d_keys_out        = init({0, 0, 0, 0, 0, 0, 0});
  std::vector<int> expected_keys_out{6, 7, 8, 3, 0, 5, 9};
  // clang-format on

  // Determine temporary device storage requirements
  void *d_temp_storage = NULL;
  size_t temp_storage_bytes = 0;
  hipcub::DeviceSegmentedRadixSort::SortKeys(
      d_temp_storage, temp_storage_bytes, d_keys_in, d_keys_out, num_items,
      num_segments, d_offsets, d_offsets + 1, 2);
  // Allocate temporary storage
  hipMalloc(&d_temp_storage, temp_storage_bytes);
  // Run sorting operation
  hipcub::DeviceSegmentedRadixSort::SortKeys(
      d_temp_storage, temp_storage_bytes, d_keys_in, d_keys_out, num_items,
      num_segments, d_offsets, d_offsets + 1, 2);

  std::vector<int> keys_out(num_items), values_out(num_items);
  hipMemcpy(keys_out.data(), d_keys_out, sizeof(int) * num_items,
             hipMemcpyDeviceToHost);
  hipFree(d_offsets);
  hipFree(d_keys_in);
  hipFree(d_keys_out);
  hipFree(d_temp_storage);

  return std::equal(keys_out.begin(), keys_out.end(),
                    expected_keys_out.begin());
}

bool test2() {
  // clang-format off
  // Declare, allocate, and initialize device-accessible pointers for sorting data
  int  num_items          = 7;
  int  num_segments       = 3;
  int  *d_offsets         = init({0, 3, 3, 7});
  int  *d_keys_in         = init({8, 6, 7, 5, 3, 0, 9});
  int  *d_keys_out        = init({0, 0, 0, 0, 0, 0, 0});
  std::vector<int> expected_keys_out{6, 7, 8, 3, 0, 5, 9};
  // clang-format on

  // Determine temporary device storage requirements
  void *d_temp_storage = NULL;
  size_t temp_storage_bytes = 0;
  hipcub::DeviceSegmentedRadixSort::SortKeys(
      d_temp_storage, temp_storage_bytes, d_keys_in, d_keys_out, num_items,
      num_segments, d_offsets, d_offsets + 1, 2, 4);
  // Allocate temporary storage
  hipMalloc(&d_temp_storage, temp_storage_bytes);
  // Run sorting operation
  hipcub::DeviceSegmentedRadixSort::SortKeys(
      d_temp_storage, temp_storage_bytes, d_keys_in, d_keys_out, num_items,
      num_segments, d_offsets, d_offsets + 1, 2, 4);

  std::vector<int> keys_out(num_items), values_out(num_items);
  hipMemcpy(keys_out.data(), d_keys_out, sizeof(int) * num_items,
             hipMemcpyDeviceToHost);

  hipFree(d_offsets);
  hipFree(d_keys_in);
  hipFree(d_keys_out);

  hipFree(d_temp_storage);

  return std::equal(keys_out.begin(), keys_out.end(),
                    expected_keys_out.begin());
}

bool test3() {
  // clang-format off
  // Declare, allocate, and initialize device-accessible pointers for sorting data
  int  num_items          = 7;
  int  num_segments       = 3;
  int  *d_offsets         = init({0, 3, 3, 7});
  int  *d_keys_in         = init({8, 6, 7, 5, 3, 0, 9});
  int  *d_keys_out        = init({0, 0, 0, 0, 0, 0, 0});
  std::vector<int> expected_keys_out{6, 7, 8, 3, 0, 5, 9};
  // clang-format on
  hipStream_t s;
  hipStreamCreate(&s);
  // Determine temporary device storage requirements
  void *d_temp_storage = NULL;
  size_t temp_storage_bytes = 0;
  hipcub::DeviceSegmentedRadixSort::SortKeys(
      d_temp_storage, temp_storage_bytes, d_keys_in, d_keys_out, num_items,
      num_segments, d_offsets, d_offsets + 1, 2, 4, s);
  // Allocate temporary storage
  hipMalloc(&d_temp_storage, temp_storage_bytes);
  // Run sorting operation
  hipcub::DeviceSegmentedRadixSort::SortKeys(
      d_temp_storage, temp_storage_bytes, d_keys_in, d_keys_out, num_items,
      num_segments, d_offsets, d_offsets + 1, 2, 4, s);

  std::vector<int> keys_out(num_items), values_out(num_items);
  hipMemcpy(keys_out.data(), d_keys_out, sizeof(int) * num_items,
             hipMemcpyDeviceToHost);
  hipFree(d_offsets);
  hipFree(d_keys_in);
  hipFree(d_keys_out);
  hipFree(d_temp_storage);
  hipStreamDestroy(s);

  return std::equal(keys_out.begin(), keys_out.end(),
                    expected_keys_out.begin());
}

bool testDescennding() {
  // clang-format off
  // Declare, allocate, and initialize device-accessible pointers for sorting data
  int  num_items          = 7;
  int  num_segments       = 3;
  int  *d_offsets         = init({0, 3, 3, 7});
  int  *d_keys_in         = init({8, 6, 7, 5, 3, 0, 9});
  int  *d_keys_out        = init({0, 0, 0, 0, 0, 0, 0});
  std::vector<int> expected_keys_out{8, 7, 6, 9, 5, 3, 0};
  // clang-format on

  // Determine temporary device storage requirements
  void *d_temp_storage = NULL;
  size_t temp_storage_bytes = 0;
  hipcub::DeviceSegmentedRadixSort::SortKeysDescending(
      d_temp_storage, temp_storage_bytes, d_keys_in, d_keys_out, num_items,
      num_segments, d_offsets, d_offsets + 1);
  // Allocate temporary storage
  hipMalloc(&d_temp_storage, temp_storage_bytes);
  // Run sorting operation
  hipcub::DeviceSegmentedRadixSort::SortKeysDescending(
      d_temp_storage, temp_storage_bytes, d_keys_in, d_keys_out, num_items,
      num_segments, d_offsets, d_offsets + 1);

  std::vector<int> keys_out(num_items), values_out(num_items);
  hipMemcpy(keys_out.data(), d_keys_out, sizeof(int) * num_items,
             hipMemcpyDeviceToHost);
  hipFree(d_offsets);
  hipFree(d_keys_in);
  hipFree(d_keys_out);
  hipFree(d_temp_storage);
  return std::equal(keys_out.begin(), keys_out.end(),
                    expected_keys_out.begin());
}

bool testDescennding1() {
  // clang-format off
  // Declare, allocate, and initialize device-accessible pointers for sorting data
  int  num_items          = 7;
  int  num_segments       = 3;
  int  *d_offsets         = init({0, 3, 3, 7});
  int  *d_keys_in         = init({8, 6, 7, 5, 3, 0, 9});
  int  *d_keys_out        = init({0, 0, 0, 0, 0, 0, 0});
  std::vector<int> expected_keys_out{8, 6, 7, 9, 5, 3, 0};
  // clang-format on

  // Determine temporary device storage requirements
  void *d_temp_storage = NULL;
  size_t temp_storage_bytes = 0;
  hipcub::DeviceSegmentedRadixSort::SortKeysDescending(
      d_temp_storage, temp_storage_bytes, d_keys_in, d_keys_out, num_items,
      num_segments, d_offsets, d_offsets + 1, 2);
  // Allocate temporary storage
  hipMalloc(&d_temp_storage, temp_storage_bytes);
  // Run sorting operation
  hipcub::DeviceSegmentedRadixSort::SortKeysDescending(
      d_temp_storage, temp_storage_bytes, d_keys_in, d_keys_out, num_items,
      num_segments, d_offsets, d_offsets + 1, 2);

  std::vector<int> keys_out(num_items), values_out(num_items);
  hipMemcpy(keys_out.data(), d_keys_out, sizeof(int) * num_items,
             hipMemcpyDeviceToHost);
  hipFree(d_offsets);
  hipFree(d_keys_in);
  hipFree(d_keys_out);
  hipFree(d_temp_storage);

  return std::equal(keys_out.begin(), keys_out.end(),
                    expected_keys_out.begin());
}

bool testDescennding2() {
  // clang-format off
  // Declare, allocate, and initialize device-accessible pointers for sorting data
  int  num_items          = 7;
  int  num_segments       = 3;
  int  *d_offsets         = init({0, 3, 3, 7});
  int  *d_keys_in         = init({8, 6, 7, 5, 3, 0, 9});
  int  *d_keys_out        = init({0, 0, 0, 0, 0, 0, 0});
  std::vector<int> expected_keys_out{8, 6, 7, 9, 5, 3, 0};
  // clang-format on

  // Determine temporary device storage requirements
  void *d_temp_storage = NULL;
  size_t temp_storage_bytes = 0;
  hipcub::DeviceSegmentedRadixSort::SortKeysDescending(
      d_temp_storage, temp_storage_bytes, d_keys_in, d_keys_out, num_items,
      num_segments, d_offsets, d_offsets + 1, 2, 4);
  // Allocate temporary storage
  hipMalloc(&d_temp_storage, temp_storage_bytes);
  // Run sorting operation
  hipcub::DeviceSegmentedRadixSort::SortKeysDescending(
      d_temp_storage, temp_storage_bytes, d_keys_in, d_keys_out, num_items,
      num_segments, d_offsets, d_offsets + 1, 2, 4);

  std::vector<int> keys_out(num_items), values_out(num_items);
  hipMemcpy(keys_out.data(), d_keys_out, sizeof(int) * num_items,
             hipMemcpyDeviceToHost);
  hipFree(d_offsets);
  hipFree(d_keys_in);
  hipFree(d_keys_out);
  hipFree(d_temp_storage);

  return std::equal(keys_out.begin(), keys_out.end(),
                    expected_keys_out.begin());
}

bool testDescennding3() {
  // clang-format off
  // Declare, allocate, and initialize device-accessible pointers for sorting data
  int  num_items          = 7;
  int  num_segments       = 3;
  int  *d_offsets         = init({0, 3, 3, 7});
  int  *d_keys_in         = init({8, 6, 7, 5, 3, 0, 9});
  int  *d_keys_out        = init({0, 0, 0, 0, 0, 0, 0});
  std::vector<int> expected_keys_out{8, 6, 7, 9, 5, 3, 0};
  // clang-format on
  hipStream_t s;
  hipStreamCreate(&s);
  // Determine temporary device storage requirements
  void *d_temp_storage = NULL;
  size_t temp_storage_bytes = 0;
  hipcub::DeviceSegmentedRadixSort::SortKeysDescending(
      d_temp_storage, temp_storage_bytes, d_keys_in, d_keys_out, num_items,
      num_segments, d_offsets, d_offsets + 1, 2, 4, s);
  // Allocate temporary storage
  hipMalloc(&d_temp_storage, temp_storage_bytes);
  // Run sorting operation
  hipcub::DeviceSegmentedRadixSort::SortKeysDescending(
      d_temp_storage, temp_storage_bytes, d_keys_in, d_keys_out, num_items,
      num_segments, d_offsets, d_offsets + 1, 2, 4, s);

  std::vector<int> keys_out(num_items), values_out(num_items);
  hipMemcpy(keys_out.data(), d_keys_out, sizeof(int) * num_items,
             hipMemcpyDeviceToHost);
  hipFree(d_offsets);
  hipFree(d_keys_in);
  hipFree(d_keys_out);
  hipFree(d_temp_storage);
  hipStreamDestroy(s);

  return std::equal(keys_out.begin(), keys_out.end(),
                    expected_keys_out.begin());
}

bool testDoubleBuffer() {
  // clang-format off
  // Declare, allocate, and initialize device-accessible pointers for sorting data
  int  num_items          = 7;
  int  num_segments       = 3;
  int  *d_offsets         = init({0, 3, 3, 7});
  int  *d_keys_in         = init({8, 6, 7, 5, 3, 0, 9});
  int  *d_keys_out        = init({0, 0, 0, 0, 0, 0, 0});
  hipcub::DoubleBuffer<int> d_keys(d_keys_in, d_keys_out);
  std::vector<int> expected_keys_out{6, 7, 8, 0, 3, 5, 9};
  // clang-format on

  // Determine temporary device storage requirements
  void *d_temp_storage = NULL;
  size_t temp_storage_bytes = 0;
  hipcub::DeviceSegmentedRadixSort::SortKeys(d_temp_storage, temp_storage_bytes,
                                          d_keys, num_items, num_segments,
                                          d_offsets, d_offsets + 1);
  // Allocate temporary storage
  hipMalloc(&d_temp_storage, temp_storage_bytes);
  // Run sorting operation
  hipcub::DeviceSegmentedRadixSort::SortKeys(d_temp_storage, temp_storage_bytes,
                                          d_keys, num_items, num_segments,
                                          d_offsets, d_offsets + 1);

  std::vector<int> keys_out(num_items), values_out(num_items);
  hipMemcpy(keys_out.data(), d_keys.Current(), sizeof(int) * num_items,
             hipMemcpyDeviceToHost);
  hipFree(d_offsets);
  hipFree(d_keys_in);
  hipFree(d_keys_out);
  hipFree(d_temp_storage);
  return std::equal(keys_out.begin(), keys_out.end(),
                    expected_keys_out.begin());
}

bool testDoubleBuffer1() {
  // clang-format off
  // Declare, allocate, and initialize device-accessible pointers for sorting data
  int  num_items          = 7;
  int  num_segments       = 3;
  int  *d_offsets         = init({0, 3, 3, 7});
  int  *d_keys_in         = init({8, 6, 7, 5, 3, 0, 9});
  int  *d_keys_out        = init({0, 0, 0, 0, 0, 0, 0});
  hipcub::DoubleBuffer<int> d_keys(d_keys_in, d_keys_out);
  std::vector<int> expected_keys_out{6, 7, 8, 3, 0, 5, 9};
  // clang-format on

  // Determine temporary device storage requirements
  void *d_temp_storage = NULL;
  size_t temp_storage_bytes = 0;
  hipcub::DeviceSegmentedRadixSort::SortKeys(d_temp_storage, temp_storage_bytes,
                                          d_keys, num_items, num_segments,
                                          d_offsets, d_offsets + 1, 2);
  // Allocate temporary storage
  hipMalloc(&d_temp_storage, temp_storage_bytes);
  // Run sorting operation
  hipcub::DeviceSegmentedRadixSort::SortKeys(d_temp_storage, temp_storage_bytes,
                                          d_keys, num_items, num_segments,
                                          d_offsets, d_offsets + 1, 2);

  std::vector<int> keys_out(num_items), values_out(num_items);
  hipMemcpy(keys_out.data(), d_keys.Current(), sizeof(int) * num_items,
             hipMemcpyDeviceToHost);
  hipFree(d_offsets);
  hipFree(d_keys_in);
  hipFree(d_keys_out);
  hipFree(d_temp_storage);

  return std::equal(keys_out.begin(), keys_out.end(),
                    expected_keys_out.begin());
}

bool testDoubleBuffer2() {
  // clang-format off
  // Declare, allocate, and initialize device-accessible pointers for sorting data
  int  num_items          = 7;
  int  num_segments       = 3;
  int  *d_offsets         = init({0, 3, 3, 7});
  int  *d_keys_in         = init({8, 6, 7, 5, 3, 0, 9});
  int  *d_keys_out        = init({0, 0, 0, 0, 0, 0, 0});
  hipcub::DoubleBuffer<int> d_keys(d_keys_in, d_keys_out);
  std::vector<int> expected_keys_out{6, 7, 8, 3, 0, 5, 9};
  // clang-format on

  // Determine temporary device storage requirements
  void *d_temp_storage = NULL;
  size_t temp_storage_bytes = 0;
  hipcub::DeviceSegmentedRadixSort::SortKeys(d_temp_storage, temp_storage_bytes,
                                          d_keys, num_items, num_segments,
                                          d_offsets, d_offsets + 1, 2, 4);
  // Allocate temporary storage
  hipMalloc(&d_temp_storage, temp_storage_bytes);
  // Run sorting operation
  hipcub::DeviceSegmentedRadixSort::SortKeys(d_temp_storage, temp_storage_bytes,
                                          d_keys, num_items, num_segments,
                                          d_offsets, d_offsets + 1, 2, 4);

  std::vector<int> keys_out(num_items), values_out(num_items);
  hipMemcpy(keys_out.data(), d_keys.Current(), sizeof(int) * num_items,
             hipMemcpyDeviceToHost);

  hipFree(d_offsets);
  hipFree(d_keys_in);
  hipFree(d_keys_out);

  hipFree(d_temp_storage);

  return std::equal(keys_out.begin(), keys_out.end(),
                    expected_keys_out.begin());
}

bool testDoubleBuffer3() {
  // clang-format off
  // Declare, allocate, and initialize device-accessible pointers for sorting data
  int  num_items          = 7;
  int  num_segments       = 3;
  int  *d_offsets         = init({0, 3, 3, 7});
  int  *d_keys_in         = init({8, 6, 7, 5, 3, 0, 9});
  int  *d_keys_out        = init({0, 0, 0, 0, 0, 0, 0});
  hipcub::DoubleBuffer<int> d_keys(d_keys_in, d_keys_out);
  std::vector<int> expected_keys_out{6, 7, 8, 3, 0, 5, 9};
  // clang-format on
  hipStream_t s;
  hipStreamCreate(&s);
  // Determine temporary device storage requirements
  void *d_temp_storage = NULL;
  size_t temp_storage_bytes = 0;
  hipcub::DeviceSegmentedRadixSort::SortKeys(d_temp_storage, temp_storage_bytes,
                                          d_keys, num_items, num_segments,
                                          d_offsets, d_offsets + 1, 2, 4, s);
  // Allocate temporary storage
  hipMalloc(&d_temp_storage, temp_storage_bytes);
  // Run sorting operation
  hipcub::DeviceSegmentedRadixSort::SortKeys(d_temp_storage, temp_storage_bytes,
                                          d_keys, num_items, num_segments,
                                          d_offsets, d_offsets + 1, 2, 4, s);

  std::vector<int> keys_out(num_items), values_out(num_items);
  hipMemcpy(keys_out.data(), d_keys.Current(), sizeof(int) * num_items,
             hipMemcpyDeviceToHost);
  hipFree(d_offsets);
  hipFree(d_keys_in);
  hipFree(d_keys_out);
  hipFree(d_temp_storage);
  hipStreamDestroy(s);

  return std::equal(keys_out.begin(), keys_out.end(),
                    expected_keys_out.begin());
}

bool testDoubleBufferDescennding() {
  // clang-format off
  // Declare, allocate, and initialize device-accessible pointers for sorting data
  int  num_items          = 7;
  int  num_segments       = 3;
  int  *d_offsets         = init({0, 3, 3, 7});
  int  *d_keys_in         = init({8, 6, 7, 5, 3, 0, 9});
  int  *d_keys_out        = init({0, 0, 0, 0, 0, 0, 0});
  hipcub::DoubleBuffer<int> d_keys(d_keys_in, d_keys_out);
  std::vector<int> expected_keys_out{8, 7, 6, 9, 5, 3, 0};
  // clang-format on

  // Determine temporary device storage requirements
  void *d_temp_storage = NULL;
  size_t temp_storage_bytes = 0;
  hipcub::DeviceSegmentedRadixSort::SortKeysDescending(
      d_temp_storage, temp_storage_bytes, d_keys, num_items, num_segments,
      d_offsets, d_offsets + 1);
  // Allocate temporary storage
  hipMalloc(&d_temp_storage, temp_storage_bytes);
  // Run sorting operation
  hipcub::DeviceSegmentedRadixSort::SortKeysDescending(
      d_temp_storage, temp_storage_bytes, d_keys, num_items, num_segments,
      d_offsets, d_offsets + 1);

  std::vector<int> keys_out(num_items), values_out(num_items);
  hipMemcpy(keys_out.data(), d_keys.Current(), sizeof(int) * num_items,
             hipMemcpyDeviceToHost);
  hipFree(d_offsets);
  hipFree(d_keys_in);
  hipFree(d_keys_out);
  hipFree(d_temp_storage);
  return std::equal(keys_out.begin(), keys_out.end(),
                    expected_keys_out.begin());
}

bool testDoubleBufferDescennding1() {
  // clang-format off
  // Declare, allocate, and initialize device-accessible pointers for sorting data
  int  num_items          = 7;
  int  num_segments       = 3;
  int  *d_offsets         = init({0, 3, 3, 7});
  int  *d_keys_in         = init({8, 6, 7, 5, 3, 0, 9});
  int  *d_keys_out        = init({0, 0, 0, 0, 0, 0, 0});
  hipcub::DoubleBuffer<int> d_keys(d_keys_in, d_keys_out);
  std::vector<int> expected_keys_out{8, 6, 7, 9, 5, 3, 0};
  // clang-format on

  // Determine temporary device storage requirements
  void *d_temp_storage = NULL;
  size_t temp_storage_bytes = 0;
  hipcub::DeviceSegmentedRadixSort::SortKeysDescending(
      d_temp_storage, temp_storage_bytes, d_keys, num_items, num_segments,
      d_offsets, d_offsets + 1, 2);
  // Allocate temporary storage
  hipMalloc(&d_temp_storage, temp_storage_bytes);
  // Run sorting operation
  hipcub::DeviceSegmentedRadixSort::SortKeysDescending(
      d_temp_storage, temp_storage_bytes, d_keys, num_items, num_segments,
      d_offsets, d_offsets + 1, 2);

  std::vector<int> keys_out(num_items), values_out(num_items);
  hipMemcpy(keys_out.data(), d_keys.Current(), sizeof(int) * num_items,
             hipMemcpyDeviceToHost);
  hipFree(d_offsets);
  hipFree(d_keys_in);
  hipFree(d_keys_out);
  hipFree(d_temp_storage);

  return std::equal(keys_out.begin(), keys_out.end(),
                    expected_keys_out.begin());
}

bool testDoubleBufferDescennding2() {
  // clang-format off
  // Declare, allocate, and initialize device-accessible pointers for sorting data
  int  num_items          = 7;
  int  num_segments       = 3;
  int  *d_offsets         = init({0, 3, 3, 7});
  int  *d_keys_in         = init({8, 6, 7, 5, 3, 0, 9});
  int  *d_keys_out        = init({0, 0, 0, 0, 0, 0, 0});
  hipcub::DoubleBuffer<int> d_keys(d_keys_in, d_keys_out);
  std::vector<int> expected_keys_out{8, 6, 7, 9, 5, 3, 0};
  // clang-format on

  // Determine temporary device storage requirements
  void *d_temp_storage = NULL;
  size_t temp_storage_bytes = 0;
  hipcub::DeviceSegmentedRadixSort::SortKeysDescending(
      d_temp_storage, temp_storage_bytes, d_keys, num_items, num_segments,
      d_offsets, d_offsets + 1, 2, 4);
  // Allocate temporary storage
  hipMalloc(&d_temp_storage, temp_storage_bytes);
  // Run sorting operation
  hipcub::DeviceSegmentedRadixSort::SortKeysDescending(
      d_temp_storage, temp_storage_bytes, d_keys, num_items, num_segments,
      d_offsets, d_offsets + 1, 2, 4);

  std::vector<int> keys_out(num_items), values_out(num_items);
  hipMemcpy(keys_out.data(), d_keys.Current(), sizeof(int) * num_items,
             hipMemcpyDeviceToHost);
  hipFree(d_offsets);
  hipFree(d_keys_in);
  hipFree(d_keys_out);
  hipFree(d_temp_storage);

  return std::equal(keys_out.begin(), keys_out.end(),
                    expected_keys_out.begin());
}

bool testDoubleBufferDescennding3() {
  // clang-format off
  // Declare, allocate, and initialize device-accessible pointers for sorting data
  int  num_items          = 7;
  int  num_segments       = 3;
  int  *d_offsets         = init({0, 3, 3, 7});
  int  *d_keys_in         = init({8, 6, 7, 5, 3, 0, 9});
  int  *d_keys_out        = init({0, 0, 0, 0, 0, 0, 0});
  hipcub::DoubleBuffer<int> d_keys(d_keys_in, d_keys_out);
  std::vector<int> expected_keys_out{8, 6, 7, 9, 5, 3, 0};
  // clang-format on
  hipStream_t s;
  hipStreamCreate(&s);
  // Determine temporary device storage requirements
  void *d_temp_storage = NULL;
  size_t temp_storage_bytes = 0;
  hipcub::DeviceSegmentedRadixSort::SortKeysDescending(
      d_temp_storage, temp_storage_bytes, d_keys, num_items, num_segments,
      d_offsets, d_offsets + 1, 2, 4, s);
  // Allocate temporary storage
  hipMalloc(&d_temp_storage, temp_storage_bytes);
  // Run sorting operation
  hipcub::DeviceSegmentedRadixSort::SortKeysDescending(
      d_temp_storage, temp_storage_bytes, d_keys, num_items, num_segments,
      d_offsets, d_offsets + 1, 2, 4, s);

  std::vector<int> keys_out(num_items), values_out(num_items);
  hipMemcpy(keys_out.data(), d_keys.Current(), sizeof(int) * num_items,
             hipMemcpyDeviceToHost);
  hipFree(d_offsets);
  hipFree(d_keys_in);
  hipFree(d_keys_out);
  hipFree(d_temp_storage);
  hipStreamDestroy(s);

  return std::equal(keys_out.begin(), keys_out.end(),
                    expected_keys_out.begin());
}

int main() {
  if (!test()) {
    printf("hipcub::DeviceSegmentedRadixSort::SortKeys test failed\n");
    return EXIT_FAILURE;
  }
  if (!test1()) {
    printf("hipcub::DeviceSegmentedRadixSort::SortKeys test1 failed\n");
    return EXIT_FAILURE;
  }
  if (!test2()) {
    printf("hipcub::DeviceSegmentedRadixSort::SortKeys test2 failed\n");
    return EXIT_FAILURE;
  }
  if (!test3()) {
    printf("hipcub::DeviceSegmentedRadixSort::SortKeys test3 failed\n");
    return EXIT_FAILURE;
  }

  if (!testDescennding()) {
    printf("hipcub::DeviceSegmentedRadixSort::SortKeysDescending testDescennding "
           "failed\n");
    return EXIT_FAILURE;
  }

  if (!testDescennding1()) {
    printf("hipcub::DeviceSegmentedRadixSort::SortKeysDescending testDescennding1 "
           "failed\n");
    return EXIT_FAILURE;
  }

  if (!testDescennding2()) {
    printf("hipcub::DeviceSegmentedRadixSort::SortKeysDescending testDescennding2 "
           "failed\n");
    return EXIT_FAILURE;
  }

  if (!testDescennding3()) {
    printf("hipcub::DeviceSegmentedRadixSort::SortKeysDescending testDescennding3 "
           "failed\n");
    return EXIT_FAILURE;
  }

  if (!testDoubleBuffer()) {
    printf("hipcub::DeviceSegmentedRadixSort::SortKeys testDoubleBuffer failed\n");
    return EXIT_FAILURE;
  }
  if (!testDoubleBuffer1()) {
    printf(
        "hipcub::DeviceSegmentedRadixSort::SortKeys testDoubleBuffer1 failed\n");
    return EXIT_FAILURE;
  }
  if (!testDoubleBuffer2()) {
    printf(
        "hipcub::DeviceSegmentedRadixSort::SortKeys testDoubleBuffer2 failed\n");
    return EXIT_FAILURE;
  }
  if (!testDoubleBuffer3()) {
    printf(
        "hipcub::DeviceSegmentedRadixSort::SortKeys testDoubleBuffer3 failed\n");
    return EXIT_FAILURE;
  }

  if (!testDoubleBufferDescennding()) {
    printf("hipcub::DeviceSegmentedRadixSort::SortKeysDescending "
           "testDoubleBufferDescennding failed\n");
    return EXIT_FAILURE;
  }

  if (!testDoubleBufferDescennding1()) {
    printf("hipcub::DeviceSegmentedRadixSort::SortKeysDescending "
           "testDoubleBufferDescennding1 failed\n");
    return EXIT_FAILURE;
  }

  if (!testDoubleBufferDescennding2()) {
    printf("hipcub::DeviceSegmentedRadixSort::SortKeysDescending "
           "testDoubleBufferDescennding2 failed\n");
    return EXIT_FAILURE;
  }

  if (!testDoubleBufferDescennding3()) {
    printf("hipcub::DeviceSegmentedRadixSort::SortKeysDescending "
           "testDoubleBufferDescennding3 failed\n");
    return EXIT_FAILURE;
  }

  return 0;
}
