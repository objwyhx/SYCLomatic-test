#include <hipcub/hipcub.hpp>
#include <initializer_list>
#include <stdio.h>
#include <vector>
#include <iostream>

template <typename T> T *init(std::initializer_list<T> list) {
  T *arr = nullptr;
  hipMalloc(&arr, sizeof(T) * list.size());
  hipMemcpy(arr, list.begin(), sizeof(T) * list.size(),
             hipMemcpyHostToDevice);
  return arr;
}

bool test(bool useDoubleBuffer=false) {
  int num_items = 7;
  int *d_keys_in = init({8, 6, 7, 5, 3, 0, 9});
  int *d_keys_out = init({0, 0, 0, 0, 0, 0, 0});
  std::vector<int> expected_keys_out{0, 3, 5, 6, 7, 8, 9};
  hipcub::DoubleBuffer<int> buffers(d_keys_in, d_keys_out);

  void *d_temp_storage = NULL;
  size_t temp_storage_bytes = 0;
  auto doSort = [&]() {
    if (useDoubleBuffer) {
      hipcub::DeviceRadixSort::SortKeys(d_temp_storage, temp_storage_bytes, buffers,
                                     num_items);
    } else {
      hipcub::DeviceRadixSort::SortKeys(d_temp_storage, temp_storage_bytes, d_keys_in,
                                     d_keys_out, num_items);
    }
  };

  // Determine temporary device storage requirements
  doSort();
  // Allocate temporary storage
  hipMalloc(&d_temp_storage, temp_storage_bytes);
  // Run sorting operation
  doSort();
  
  std::vector<int> keys_out(num_items);
  if (useDoubleBuffer) {
    hipMemcpy(keys_out.data(), buffers.Current(), sizeof(int) * num_items,
               hipMemcpyDeviceToHost);
  } else {
    hipMemcpy(keys_out.data(), d_keys_out, sizeof(int) * num_items,
               hipMemcpyDeviceToHost);
  }
  hipFree(d_keys_in);
  hipFree(d_keys_out);
  hipFree(d_temp_storage);
  return std::equal(expected_keys_out.begin(), expected_keys_out.end(),
                    keys_out.begin());
}

bool test1(bool useDoubleBuffer=false) {
  int num_items = 7;
  int *d_keys_in = init({8, 6, 7, 5, 3, 0, 9});
  int *d_keys_out = init({0, 0, 0, 0, 0, 0, 0});
  std::vector<int> expected_keys_out{3, 0, 6, 7, 5, 8, 9};
  hipcub::DoubleBuffer<int> buffers(d_keys_in, d_keys_out);

  void *d_temp_storage = NULL;
  size_t temp_storage_bytes = 0;
  auto doSort = [&]() {
    if (useDoubleBuffer) {
      hipcub::DeviceRadixSort::SortKeys(d_temp_storage, temp_storage_bytes, buffers,
                                     num_items, 2);
    } else {
      hipcub::DeviceRadixSort::SortKeys(d_temp_storage, temp_storage_bytes, d_keys_in,
                                     d_keys_out, num_items, 2);
    }
  };

  // Determine temporary device storage requirements
  doSort();
  // Allocate temporary storage
  hipMalloc(&d_temp_storage, temp_storage_bytes);
  // Run sorting operation
  doSort();
  
  std::vector<int> keys_out(num_items);
  if (useDoubleBuffer) {
    hipMemcpy(keys_out.data(), buffers.Current(), sizeof(int) * num_items,
               hipMemcpyDeviceToHost);
  } else {
    hipMemcpy(keys_out.data(), d_keys_out, sizeof(int) * num_items,
               hipMemcpyDeviceToHost);
  }
  hipFree(d_keys_in);
  hipFree(d_keys_out);
  hipFree(d_temp_storage);
  
  return std::equal(expected_keys_out.begin(), expected_keys_out.end(),
                    keys_out.begin());
}

bool test2(bool useDoubleBuffer=false) {
  int num_items = 7;
  int *d_keys_in = init({8, 6, 7, 5, 3, 0, 9});
  int *d_keys_out = init({0, 0, 0, 0, 0, 0, 0});
  std::vector<int> expected_keys_out{3, 0, 6, 7, 5, 8, 9};
  hipcub::DoubleBuffer<int> buffers(d_keys_in, d_keys_out);

  void *d_temp_storage = NULL;
  size_t temp_storage_bytes = 0;
  auto doSort = [&]() {
    if (useDoubleBuffer) {
      hipcub::DeviceRadixSort::SortKeys(d_temp_storage, temp_storage_bytes, buffers,
                                     num_items, 2, 4);
    } else {
      hipcub::DeviceRadixSort::SortKeys(d_temp_storage, temp_storage_bytes, d_keys_in,
                                     d_keys_out, num_items, 2, 4);
    }
  };
  
  // Determine temporary device storage requirements
  doSort();
  // Allocate temporary storage
  hipMalloc(&d_temp_storage, temp_storage_bytes);
  // Run sorting operation
  doSort();
  
  std::vector<int> keys_out(num_items);
  hipMemcpy(keys_out.data(), d_keys_out, sizeof(int) * num_items,
             hipMemcpyDeviceToHost);
  if (useDoubleBuffer) {
    hipMemcpy(keys_out.data(), buffers.Current(), sizeof(int) * num_items,
               hipMemcpyDeviceToHost);
  } else {
    hipMemcpy(keys_out.data(), d_keys_out, sizeof(int) * num_items,
               hipMemcpyDeviceToHost);
  }
  hipFree(d_keys_in);
  hipFree(d_keys_out);
  hipFree(d_temp_storage);
  
  return std::equal(expected_keys_out.begin(), expected_keys_out.end(),
                    keys_out.begin());
}

bool test3(bool useDoubleBuffer=false) {
  int num_items = 7;
  int *d_keys_in = init({8, 6, 7, 5, 3, 0, 9});
  int *d_keys_out = init({0, 0, 0, 0, 0, 0, 0});
  std::vector<int> expected_keys_out{3, 0, 6, 7, 5, 8, 9};
  hipcub::DoubleBuffer<int> buffers(d_keys_in, d_keys_out);
  hipStream_t s = nullptr;
  hipStreamCreate(&s);

  void *d_temp_storage = NULL;
  size_t temp_storage_bytes = 0;
  auto doSort = [&]() {
    if (useDoubleBuffer) {
      hipcub::DeviceRadixSort::SortKeys(d_temp_storage, temp_storage_bytes, buffers,
                                     num_items, 2, 4, s);
    } else {
      hipcub::DeviceRadixSort::SortKeys(d_temp_storage, temp_storage_bytes, d_keys_in,
                                     d_keys_out, num_items, 2, 4, s);
    }
  };
  // Determine temporary device storage requirements
  doSort();
  // Allocate temporary storage
  hipMalloc(&d_temp_storage, temp_storage_bytes);
  // Run sorting operation
  doSort();

  std::vector<int> keys_out(num_items);
  if (useDoubleBuffer) {
    hipMemcpy(keys_out.data(), buffers.Current(), sizeof(int) * num_items,
               hipMemcpyDeviceToHost);
  } else {
    hipMemcpy(keys_out.data(), d_keys_out, sizeof(int) * num_items,
               hipMemcpyDeviceToHost);
  }
  hipFree(d_keys_in);
  hipFree(d_keys_out);
  hipFree(d_temp_storage);
  hipStreamDestroy(s);
  
  return std::equal(expected_keys_out.begin(), expected_keys_out.end(),
                    keys_out.begin());
}

bool test_descending(bool useDoubleBuffer=false) {
  int num_items = 7;
  int *d_keys_in = init({8, 6, 7, 5, 3, 0, 9});
  int *d_keys_out = init({0, 0, 0, 0, 0, 0, 0});
  std::vector<int> expected_keys_out{9, 8, 7, 6, 5, 3, 0};
  hipcub::DoubleBuffer<int> buffers(d_keys_in, d_keys_out);

  void *d_temp_storage = NULL;
  size_t temp_storage_bytes = 0;
  auto doSort = [&]() {
    if (useDoubleBuffer) {
      hipcub::DeviceRadixSort::SortKeysDescending(d_temp_storage, temp_storage_bytes,
                                               buffers, num_items);
    } else {
      hipcub::DeviceRadixSort::SortKeysDescending(d_temp_storage, temp_storage_bytes,
                                               d_keys_in, d_keys_out, num_items);
    }
  };

  // Determine temporary device storage requirements
  doSort();
  // Allocate temporary storage
  hipMalloc(&d_temp_storage, temp_storage_bytes);
  // Run sorting operation
  doSort();

  std::vector<int> keys_out(num_items);
  if (useDoubleBuffer) {
    hipMemcpy(keys_out.data(), buffers.Current(), sizeof(int) * num_items,
               hipMemcpyDeviceToHost);
  } else {
    hipMemcpy(keys_out.data(), d_keys_out, sizeof(int) * num_items,
               hipMemcpyDeviceToHost);
  }
  hipFree(d_keys_in);
  hipFree(d_keys_out);
  hipFree(d_temp_storage);
  return std::equal(expected_keys_out.begin(), expected_keys_out.end(),
                    keys_out.begin());
}

bool test_descending1(bool useDoubleBuffer=false) {
  int num_items = 7;
  int *d_keys_in = init({8, 6, 7, 5, 3, 0, 9});
  int *d_keys_out = init({0, 0, 0, 0, 0, 0, 0});
  std::vector<int> expected_keys_out{8, 9, 6, 7, 5, 3, 0};
  hipcub::DoubleBuffer<int> buffers(d_keys_in, d_keys_out);

  void *d_temp_storage = NULL;
  size_t temp_storage_bytes = 0;
  auto doSort = [&]() {
    if (useDoubleBuffer) {
      hipcub::DeviceRadixSort::SortKeysDescending(d_temp_storage, temp_storage_bytes,
                                               buffers, num_items, 2);
    } else {
      hipcub::DeviceRadixSort::SortKeysDescending(d_temp_storage, temp_storage_bytes,
                                               d_keys_in, d_keys_out, num_items, 2);
    }
  };
  // Determine temporary device storage requirements
  doSort();
  // Allocate temporary storage
  hipMalloc(&d_temp_storage, temp_storage_bytes);
  // Run sorting operation
  doSort();

  std::vector<int> keys_out(num_items);
  if (useDoubleBuffer) {
    hipMemcpy(keys_out.data(), buffers.Current(), sizeof(int) * num_items,
               hipMemcpyDeviceToHost);
  } else {
    hipMemcpy(keys_out.data(), d_keys_out, sizeof(int) * num_items,
               hipMemcpyDeviceToHost);
  }
  hipFree(d_keys_in);
  hipFree(d_keys_out);
  hipFree(d_temp_storage);
  
  return std::equal(expected_keys_out.begin(), expected_keys_out.end(),
                    keys_out.begin());
}

bool test_descending2(bool useDoubleBuffer=false) {
  int num_items = 7;
  int *d_keys_in = init({8, 6, 7, 5, 3, 0, 9});
  int *d_keys_out = init({0, 0, 0, 0, 0, 0, 0});
  std::vector<int> expected_keys_out{8, 9, 6, 7, 5, 3, 0};
  hipcub::DoubleBuffer<int> buffers(d_keys_in, d_keys_out);

  void *d_temp_storage = NULL;
  size_t temp_storage_bytes = 0;
  auto doSort = [&]() {
    if (useDoubleBuffer) {
      hipcub::DeviceRadixSort::SortKeysDescending(d_temp_storage, temp_storage_bytes,
                                               buffers, num_items, 2, 4);
    } else {
      hipcub::DeviceRadixSort::SortKeysDescending(d_temp_storage, temp_storage_bytes,
                                               d_keys_in, d_keys_out, num_items, 2, 4);
    }
  };

  // Determine temporary device storage requirements
  doSort();
  // Allocate temporary storage
  hipMalloc(&d_temp_storage, temp_storage_bytes);
  // Run sorting operation
  doSort();

  std::vector<int> keys_out(num_items);
  if (useDoubleBuffer) {
    hipMemcpy(keys_out.data(), buffers.Current(), sizeof(int) * num_items,
               hipMemcpyDeviceToHost);
  } else {
    hipMemcpy(keys_out.data(), d_keys_out, sizeof(int) * num_items,
               hipMemcpyDeviceToHost);
  }
  hipFree(d_keys_in);
  hipFree(d_keys_out);
  hipFree(d_temp_storage);
  
  return std::equal(expected_keys_out.begin(), expected_keys_out.end(),
                    keys_out.begin());
}

bool test_descending3(bool useDoubleBuffer=false) {
  int num_items = 7;
  int *d_keys_in = init({8, 6, 7, 5, 3, 0, 9});
  int *d_keys_out = init({0, 0, 0, 0, 0, 0, 0});
  std::vector<int> expected_keys_out{8, 9, 6, 7, 5, 3, 0};
  hipStream_t s = nullptr;
  hipStreamCreate(&s);
  hipcub::DoubleBuffer<int> buffers(d_keys_in, d_keys_out);

  void *d_temp_storage = NULL;
  size_t temp_storage_bytes = 0;
  auto doSort = [&]() {
    if (useDoubleBuffer) {
      hipcub::DeviceRadixSort::SortKeysDescending(d_temp_storage, temp_storage_bytes,
                                               buffers, num_items, 2, 4, s);
    } else {
      hipcub::DeviceRadixSort::SortKeysDescending(d_temp_storage, temp_storage_bytes,
                                               d_keys_in, d_keys_out, num_items, 2, 4, s);
    }
  };
  // Determine temporary device storage requirements
  doSort();
  // Allocate temporary storage
  hipMalloc(&d_temp_storage, temp_storage_bytes);
  // Run sorting operation
  doSort();

  std::vector<int> keys_out(num_items);
  if (useDoubleBuffer) {
    hipMemcpy(keys_out.data(), buffers.Current(), sizeof(int) * num_items,
               hipMemcpyDeviceToHost);
  } else {
    hipMemcpy(keys_out.data(), d_keys_out, sizeof(int) * num_items,
               hipMemcpyDeviceToHost);
  }
  hipFree(d_keys_in);
  hipFree(d_keys_out);
  hipFree(d_temp_storage);
  hipStreamDestroy(s);
  
  return std::equal(expected_keys_out.begin(), expected_keys_out.end(),
                    keys_out.begin());
}

int main() {
  int res = 0;
  for (auto b : {false, true}) {
    auto s = b ? " with double buffer" : "";
    if (!test(b)) {
      printf("hipcub::DeviceRadixSort::SortKeys%s failed\n", s);
      res = 1;
    }
    if (!test1(b)) {
      printf("hipcub::DeviceRadixSort::SortKeys%s failed\n", s);
      res = 1;
    }
    if (!test2(b)) {
      printf("hipcub::DeviceRadixSort::SortKeys%s failed\n", s);
      res = 1;
    }
    if (!test3(b)) {
      printf("hipcub::DeviceRadixSort::SortKeys%s failed\n", s);
      res = 1;
    }

    if (!test_descending(b)) {
      printf("hipcub::DeviceRadixSort::SortKeysDescending%s failed\n", s);
      res = 1;
    }

    if (!test_descending1(b)) {
      printf("hipcub::DeviceRadixSort::SortKeysDescending%s failed\n", s);
      res = 1;
    }

    if (!test_descending2(b)) {
      printf("hipcub::DeviceRadixSort::SortKeysDescending%s failed\n", s);
      res = 1;
    }

    if (!test_descending3(b)) {
      printf("hipcub::DeviceRadixSort::SortKeysDescending%s failed\n", s);
      res = 1;
    }
  }

  return res;
}
