//====---------- math-emu-bf16.cu ----------- *- CUDA -* ------------------===//
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
//
//===----------------------------------------------------------------------===//


#include <hip/hip_runtime.h>
#include <iomanip>
#include <iostream>
#include <vector>

#include "hip/hip_bf16.h"

using namespace std;

typedef vector<__hip_bfloat16> bf16_vector;
typedef pair<__hip_bfloat16, int> bf16i_pair;

int passed = 0;
int failed = 0;

void check(bool IsPassed) {
  if (IsPassed) {
    cout << " ---- passed" << endl;
    passed++;
  } else {
    cout << " ---- failed" << endl;
    failed++;
  }
}

void checkResult(const string &FuncName, const vector<float> &Inputs,
                 const float &Expect, const float &Result,
                 const int precision) {
  cout << FuncName << "(" << Inputs[0];
  for (size_t i = 1; i < Inputs.size(); ++i) {
    cout << ", " << Inputs[i];
  }
  cout << ") = " << fixed << setprecision(precision < 0 ? 0 : precision)
       << Result << " (expect " << Expect - pow(10, -precision) << " ~ "
       << Expect + pow(10, -precision) << ")";
  cout.unsetf(ios::fixed);
  check(abs(Result - Expect) < pow(10, -precision));
}

void checkResult(const string &FuncName, const vector<float> &Inputs,
                 const bool &Expect, const bool &Result) {
  cout << FuncName << "(" << Inputs[0];
  for (size_t i = 1; i < Inputs.size(); ++i) {
    cout << ", " << Inputs[i];
  }
  cout << ") = " << Result << " (expect " << Expect << ")";
  check(Result == Expect);
}

void checkResult(const string &FuncName, const vector<__hip_bfloat16> &Inputs,
                 const __hip_bfloat16 &Expect, const float &Result,
                 const int precision) {
  vector<float> FInputs;
  for (const auto &it : Inputs) {
    FInputs.push_back(__bfloat162float(it));
  }
  float FExpect{__bfloat162float(Expect)};
  checkResult(FuncName, FInputs, FExpect, Result, precision);
}

// Bfloat16 Arithmetic Functions

__global__ void habs(float *const Result, __hip_bfloat16 Input1) {
  *Result = __habs(Input1);
}

void testHabsCases(const vector<pair<__hip_bfloat16, bf16i_pair>> &TestCases) {
  float *Result;
  hipMallocManaged(&Result, sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    habs<<<1, 1>>>(Result, TestCase.first);
    hipDeviceSynchronize();
    checkResult("__habs", {TestCase.first}, TestCase.second.first, *Result,
                TestCase.second.second);
  }
}

__global__ void hadd(float *const Result, __hip_bfloat16 Input1,
                     __hip_bfloat16 Input2) {
  *Result = __hadd(Input1, Input2);
}

void testHaddCases(
    const vector<pair<pair<__hip_bfloat16, __hip_bfloat16>, bf16i_pair>>
        &TestCases) {
  float *Result;
  hipMallocManaged(&Result, sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    hadd<<<1, 1>>>(Result, TestCase.first.first, TestCase.first.second);
    hipDeviceSynchronize();
    checkResult("__hadd", {TestCase.first.first, TestCase.first.second},
                TestCase.second.first, *Result, TestCase.second.second);
  }
}

__global__ void hadd_rn(float *const Result, __hip_bfloat16 Input1,
                        __hip_bfloat16 Input2) {
  *Result = __hadd_rn(Input1, Input2);
}

void testHadd_rnCases(
    const vector<pair<pair<__hip_bfloat16, __hip_bfloat16>, bf16i_pair>>
        &TestCases) {
  float *Result;
  hipMallocManaged(&Result, sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    hadd_rn<<<1, 1>>>(Result, TestCase.first.first, TestCase.first.second);
    hipDeviceSynchronize();
    checkResult("__hadd_rn", {TestCase.first.first, TestCase.first.second},
                TestCase.second.first, *Result, TestCase.second.second);
  }
}

__global__ void hadd_sat(float *const Result, __hip_bfloat16 Input1,
                         __hip_bfloat16 Input2) {
  *Result = __hadd_sat(Input1, Input2);
}

void testHadd_satCases(
    const vector<pair<pair<__hip_bfloat16, __hip_bfloat16>, bf16i_pair>>
        &TestCases) {
  float *Result;
  hipMallocManaged(&Result, sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    hadd_sat<<<1, 1>>>(Result, TestCase.first.first, TestCase.first.second);
    hipDeviceSynchronize();
    checkResult("__hadd_sat", {TestCase.first.first, TestCase.first.second},
                TestCase.second.first, *Result, TestCase.second.second);
  }
}

__global__ void hdiv(float *const Result, __hip_bfloat16 Input1,
                     __hip_bfloat16 Input2) {
  *Result = __hdiv(Input1, Input2);
}

void testHdivCases(
    const vector<pair<pair<__hip_bfloat16, __hip_bfloat16>, bf16i_pair>>
        &TestCases) {
  float *Result;
  hipMallocManaged(&Result, sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    hdiv<<<1, 1>>>(Result, TestCase.first.first, TestCase.first.second);
    hipDeviceSynchronize();
    checkResult("__hdiv", {TestCase.first.first, TestCase.first.second},
                TestCase.second.first, *Result, TestCase.second.second);
  }
}

__global__ void hfma(float *const Result, __hip_bfloat16 Input1,
                     __hip_bfloat16 Input2, __hip_bfloat16 Input3) {
  *Result = __hfma(Input1, Input2, Input3);
}

void testHfmaCases(const vector<pair<bf16_vector, bf16i_pair>> &TestCases) {
  float *Result;
  hipMallocManaged(&Result, sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    hfma<<<1, 1>>>(Result, TestCase.first[0], TestCase.first[1],
                   TestCase.first[2]);
    hipDeviceSynchronize();
    checkResult("__hfma", TestCase.first, TestCase.second.first, *Result,
                TestCase.second.second);
    if (TestCase.first.size() != 3) {
      failed++;
      cout << " ---- failed" << endl;
      return;
    }
  }
}

__global__ void hfma_sat(float *const Result, __hip_bfloat16 Input1,
                         __hip_bfloat16 Input2, __hip_bfloat16 Input3) {
  *Result = __hfma_sat(Input1, Input2, Input3);
}

void testHfma_satCases(const vector<pair<bf16_vector, bf16i_pair>> &TestCases) {
  float *Result;
  hipMallocManaged(&Result, sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    hfma_sat<<<1, 1>>>(Result, TestCase.first[0], TestCase.first[1],
                       TestCase.first[2]);
    hipDeviceSynchronize();
    checkResult("__hfma_sat", TestCase.first, TestCase.second.first, *Result,
                TestCase.second.second);
    if (TestCase.first.size() != 3) {
      failed++;
      cout << " ---- failed" << endl;
      return;
    }
  }
}

__global__ void hmul(float *const Result, __hip_bfloat16 Input1,
                     __hip_bfloat16 Input2) {
  *Result = __hmul(Input1, Input2);
}

void testHmulCases(
    const vector<pair<pair<__hip_bfloat16, __hip_bfloat16>, bf16i_pair>>
        &TestCases) {
  float *Result;
  hipMallocManaged(&Result, sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    hmul<<<1, 1>>>(Result, TestCase.first.first, TestCase.first.second);
    hipDeviceSynchronize();
    checkResult("__hmul", {TestCase.first.first, TestCase.first.second},
                TestCase.second.first, *Result, TestCase.second.second);
  }
}

__global__ void hmul_rn(float *const Result, __hip_bfloat16 Input1,
                        __hip_bfloat16 Input2) {
  *Result = __hmul_rn(Input1, Input2);
}

void testHmul_rnCases(
    const vector<pair<pair<__hip_bfloat16, __hip_bfloat16>, bf16i_pair>>
        &TestCases) {
  float *Result;
  hipMallocManaged(&Result, sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    hmul_rn<<<1, 1>>>(Result, TestCase.first.first, TestCase.first.second);
    hipDeviceSynchronize();
    checkResult("__hmul_rn", {TestCase.first.first, TestCase.first.second},
                TestCase.second.first, *Result, TestCase.second.second);
  }
}

__global__ void hmul_sat(float *const Result, __hip_bfloat16 Input1,
                         __hip_bfloat16 Input2) {
  *Result = __hmul_sat(Input1, Input2);
}

void testHmul_satCases(
    const vector<pair<pair<__hip_bfloat16, __hip_bfloat16>, bf16i_pair>>
        &TestCases) {
  float *Result;
  hipMallocManaged(&Result, sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    hmul_sat<<<1, 1>>>(Result, TestCase.first.first, TestCase.first.second);
    hipDeviceSynchronize();
    checkResult("__hmul_sat", {TestCase.first.first, TestCase.first.second},
                TestCase.second.first, *Result, TestCase.second.second);
  }
}

__global__ void hneg(float *const Result, __hip_bfloat16 Input1) {
  *Result = __hneg(Input1);
}

void testHnegCases(const vector<pair<__hip_bfloat16, bf16i_pair>> &TestCases) {
  float *Result;
  hipMallocManaged(&Result, sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    hneg<<<1, 1>>>(Result, TestCase.first);
    hipDeviceSynchronize();
    checkResult("__hneg", {TestCase.first}, TestCase.second.first, *Result,
                TestCase.second.second);
  }
}

__global__ void hsub(float *const Result, __hip_bfloat16 Input1,
                     __hip_bfloat16 Input2) {
  *Result = __hsub(Input1, Input2);
}

void testHsubCases(
    const vector<pair<pair<__hip_bfloat16, __hip_bfloat16>, bf16i_pair>>
        &TestCases) {
  float *Result;
  hipMallocManaged(&Result, sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    hsub<<<1, 1>>>(Result, TestCase.first.first, TestCase.first.second);
    hipDeviceSynchronize();
    checkResult("__hsub", {TestCase.first.first, TestCase.first.second},
                TestCase.second.first, *Result, TestCase.second.second);
  }
}

__global__ void hsub_rn(float *const Result, __hip_bfloat16 Input1,
                        __hip_bfloat16 Input2) {
  *Result = __hsub_rn(Input1, Input2);
}

void testHsub_rnCases(
    const vector<pair<pair<__hip_bfloat16, __hip_bfloat16>, bf16i_pair>>
        &TestCases) {
  float *Result;
  hipMallocManaged(&Result, sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    hsub_rn<<<1, 1>>>(Result, TestCase.first.first, TestCase.first.second);
    hipDeviceSynchronize();
    checkResult("__hsub_rn", {TestCase.first.first, TestCase.first.second},
                TestCase.second.first, *Result, TestCase.second.second);
  }
}

__global__ void hsub_sat(float *const Result, __hip_bfloat16 Input1,
                         __hip_bfloat16 Input2) {
  *Result = __hsub_sat(Input1, Input2);
}

void testHsub_satCases(
    const vector<pair<pair<__hip_bfloat16, __hip_bfloat16>, bf16i_pair>>
        &TestCases) {
  float *Result;
  hipMallocManaged(&Result, sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    hsub_sat<<<1, 1>>>(Result, TestCase.first.first, TestCase.first.second);
    hipDeviceSynchronize();
    checkResult("__hsub_sat", {TestCase.first.first, TestCase.first.second},
                TestCase.second.first, *Result, TestCase.second.second);
  }
}

// Bfloat16 Comparison Functions

__global__ void heq(bool *const Result, __hip_bfloat16 Input1,
                    __hip_bfloat16 Input2) {
  *Result = __heq(Input1, Input2);
}

void testHeqCases(
    const vector<pair<pair<__hip_bfloat16, __hip_bfloat16>, bool>> &TestCases) {
  bool *Result;
  hipMallocManaged(&Result, sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    heq<<<1, 1>>>(Result, TestCase.first.first, TestCase.first.second);
    hipDeviceSynchronize();
    checkResult("__heq", {TestCase.first.first, TestCase.first.second},
                TestCase.second, *Result);
  }
}

__global__ void hequ(bool *const Result, __hip_bfloat16 Input1,
                     __hip_bfloat16 Input2) {
  *Result = __hequ(Input1, Input2);
}

void testHequCases(
    const vector<pair<pair<__hip_bfloat16, __hip_bfloat16>, bool>> &TestCases) {
  bool *Result;
  hipMallocManaged(&Result, sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    hequ<<<1, 1>>>(Result, TestCase.first.first, TestCase.first.second);
    hipDeviceSynchronize();
    checkResult("__hequ", {TestCase.first.first, TestCase.first.second},
                TestCase.second, *Result);
  }
}

__global__ void hge(bool *const Result, __hip_bfloat16 Input1,
                    __hip_bfloat16 Input2) {
  *Result = __hge(Input1, Input2);
}

void testHgeCases(
    const vector<pair<pair<__hip_bfloat16, __hip_bfloat16>, bool>> &TestCases) {
  bool *Result;
  hipMallocManaged(&Result, sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    hge<<<1, 1>>>(Result, TestCase.first.first, TestCase.first.second);
    hipDeviceSynchronize();
    checkResult("__hge", {TestCase.first.first, TestCase.first.second},
                TestCase.second, *Result);
  }
}

__global__ void hgeu(bool *const Result, __hip_bfloat16 Input1,
                     __hip_bfloat16 Input2) {
  *Result = __hgeu(Input1, Input2);
}

void testHgeuCases(
    const vector<pair<pair<__hip_bfloat16, __hip_bfloat16>, bool>> &TestCases) {
  bool *Result;
  hipMallocManaged(&Result, sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    hgeu<<<1, 1>>>(Result, TestCase.first.first, TestCase.first.second);
    hipDeviceSynchronize();
    checkResult("__hgeu", {TestCase.first.first, TestCase.first.second},
                TestCase.second, *Result);
  }
}

__global__ void hgt(bool *const Result, __hip_bfloat16 Input1,
                    __hip_bfloat16 Input2) {
  *Result = __hgt(Input1, Input2);
}

void testHgtCases(
    const vector<pair<pair<__hip_bfloat16, __hip_bfloat16>, bool>> &TestCases) {
  bool *Result;
  hipMallocManaged(&Result, sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    hgt<<<1, 1>>>(Result, TestCase.first.first, TestCase.first.second);
    hipDeviceSynchronize();
    checkResult("__hgt", {TestCase.first.first, TestCase.first.second},
                TestCase.second, *Result);
  }
}

__global__ void hgtu(bool *const Result, __hip_bfloat16 Input1,
                     __hip_bfloat16 Input2) {
  *Result = __hgtu(Input1, Input2);
}

void testHgtuCases(
    const vector<pair<pair<__hip_bfloat16, __hip_bfloat16>, bool>> &TestCases) {
  bool *Result;
  hipMallocManaged(&Result, sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    hgtu<<<1, 1>>>(Result, TestCase.first.first, TestCase.first.second);
    hipDeviceSynchronize();
    checkResult("__hgtu", {TestCase.first.first, TestCase.first.second},
                TestCase.second, *Result);
  }
}

__global__ void hisinf(bool *const Result, __hip_bfloat16 Input1) {
  *Result = __hisinf(Input1);
}

void testHisinfCases(const vector<pair<__hip_bfloat16, bool>> &TestCases) {
  bool *Result;
  hipMallocManaged(&Result, sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    hisinf<<<1, 1>>>(Result, TestCase.first);
    hipDeviceSynchronize();
    checkResult("__hisinf", {TestCase.first}, TestCase.second, *Result);
  }
}

__global__ void hisnan(bool *const Result, __hip_bfloat16 Input1) {
  *Result = __hisnan(Input1);
}

void testHisnanCases(const vector<pair<__hip_bfloat16, bool>> &TestCases) {
  bool *Result;
  hipMallocManaged(&Result, sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    hisnan<<<1, 1>>>(Result, TestCase.first);
    hipDeviceSynchronize();
    checkResult("__hisnan", {TestCase.first}, TestCase.second, *Result);
  }
}

__global__ void hle(bool *const Result, __hip_bfloat16 Input1,
                    __hip_bfloat16 Input2) {
  *Result = __hle(Input1, Input2);
}

void testHleCases(
    const vector<pair<pair<__hip_bfloat16, __hip_bfloat16>, bool>> &TestCases) {
  bool *Result;
  hipMallocManaged(&Result, sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    hle<<<1, 1>>>(Result, TestCase.first.first, TestCase.first.second);
    hipDeviceSynchronize();
    checkResult("__hle", {TestCase.first.first, TestCase.first.second},
                TestCase.second, *Result);
  }
}

__global__ void hleu(bool *const Result, __hip_bfloat16 Input1,
                     __hip_bfloat16 Input2) {
  *Result = __hleu(Input1, Input2);
}

void testHleuCases(
    const vector<pair<pair<__hip_bfloat16, __hip_bfloat16>, bool>> &TestCases) {
  bool *Result;
  hipMallocManaged(&Result, sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    hleu<<<1, 1>>>(Result, TestCase.first.first, TestCase.first.second);
    hipDeviceSynchronize();
    checkResult("__hleu", {TestCase.first.first, TestCase.first.second},
                TestCase.second, *Result);
  }
}

__global__ void hlt(bool *const Result, __hip_bfloat16 Input1,
                    __hip_bfloat16 Input2) {
  *Result = __hlt(Input1, Input2);
}

void testHltCases(
    const vector<pair<pair<__hip_bfloat16, __hip_bfloat16>, bool>> &TestCases) {
  bool *Result;
  hipMallocManaged(&Result, sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    hlt<<<1, 1>>>(Result, TestCase.first.first, TestCase.first.second);
    hipDeviceSynchronize();
    checkResult("__hlt", {TestCase.first.first, TestCase.first.second},
                TestCase.second, *Result);
  }
}

__global__ void hltu(bool *const Result, __hip_bfloat16 Input1,
                     __hip_bfloat16 Input2) {
  *Result = __hltu(Input1, Input2);
}

void testHltuCases(
    const vector<pair<pair<__hip_bfloat16, __hip_bfloat16>, bool>> &TestCases) {
  bool *Result;
  hipMallocManaged(&Result, sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    hltu<<<1, 1>>>(Result, TestCase.first.first, TestCase.first.second);
    hipDeviceSynchronize();
    checkResult("__hltu", {TestCase.first.first, TestCase.first.second},
                TestCase.second, *Result);
  }
}

__global__ void hmax(float *const Result, __hip_bfloat16 Input1,
                     __hip_bfloat16 Input2) {
  *Result = __hmax(Input1, Input2);
}

void testHmaxCases(
    const vector<pair<pair<__hip_bfloat16, __hip_bfloat16>, bf16i_pair>>
        &TestCases) {
  float *Result;
  hipMallocManaged(&Result, sizeof(*Result));
  // Boundary values.
  hmax<<<1, 1>>>(Result, NAN, NAN);
  hipDeviceSynchronize();
  cout << "__hmax(nan, nan) = " << *Result << " (expect nan)";
  check(isnan(*Result));
  for (const auto &TestCase : TestCases) {
    hmax<<<1, 1>>>(Result, TestCase.first.first, TestCase.first.second);
    hipDeviceSynchronize();
    checkResult("__hmax", {TestCase.first.first, TestCase.first.second},
                TestCase.second.first, *Result, TestCase.second.second);
  }
}

__global__ void hmax_nan(float *const Result, __hip_bfloat16 Input1,
                         __hip_bfloat16 Input2) {
  *Result = __hmax_nan(Input1, Input2);
}

void testHmax_nanCases(
    const vector<pair<pair<__hip_bfloat16, __hip_bfloat16>, bf16i_pair>>
        &TestCases) {
  float *Result;
  hipMallocManaged(&Result, sizeof(*Result));
  // Boundary values.
  hmax_nan<<<1, 1>>>(Result, NAN, NAN);
  hipDeviceSynchronize();
  cout << "__hmax_nan(nan, nan) = " << *Result << " (expect nan)";
  check(isnan(*Result));
  hmax_nan<<<1, 1>>>(Result, NAN, 1);
  hipDeviceSynchronize();
  cout << "__hmax_nan(nan, 1) = " << *Result << " (expect nan)";
  check(isnan(*Result));
  hmax_nan<<<1, 1>>>(Result, 1, NAN);
  hipDeviceSynchronize();
  cout << "__hmax_nan(1, nan) = " << *Result << " (expect nan)";
  check(isnan(*Result));
  for (const auto &TestCase : TestCases) {
    hmax_nan<<<1, 1>>>(Result, TestCase.first.first, TestCase.first.second);
    hipDeviceSynchronize();
    checkResult("__hmax_nan", {TestCase.first.first, TestCase.first.second},
                TestCase.second.first, *Result, TestCase.second.second);
  }
}

__global__ void hmin(float *const Result, __hip_bfloat16 Input1,
                     __hip_bfloat16 Input2) {
  *Result = __hmin(Input1, Input2);
}

void testHminCases(
    const vector<pair<pair<__hip_bfloat16, __hip_bfloat16>, bf16i_pair>>
        &TestCases) {
  float *Result;
  hipMallocManaged(&Result, sizeof(*Result));
  // Boundary values.
  hmin<<<1, 1>>>(Result, NAN, NAN);
  hipDeviceSynchronize();
  cout << "__hmin(nan, nan) = " << *Result << " (expect nan)";
  check(isnan(*Result));
  for (const auto &TestCase : TestCases) {
    hmin<<<1, 1>>>(Result, TestCase.first.first, TestCase.first.second);
    hipDeviceSynchronize();
    checkResult("__hmin", {TestCase.first.first, TestCase.first.second},
                TestCase.second.first, *Result, TestCase.second.second);
  }
}

__global__ void hmin_nan(float *const Result, __hip_bfloat16 Input1,
                         __hip_bfloat16 Input2) {
  *Result = __hmin_nan(Input1, Input2);
}

void testHmin_nanCases(
    const vector<pair<pair<__hip_bfloat16, __hip_bfloat16>, bf16i_pair>>
        &TestCases) {
  float *Result;
  hipMallocManaged(&Result, sizeof(*Result));
  // Boundary values.
  hmin_nan<<<1, 1>>>(Result, NAN, NAN);
  hipDeviceSynchronize();
  cout << "__hmin_nan(nan, nan) = " << *Result << " (expect nan)";
  check(isnan(*Result));
  hmin_nan<<<1, 1>>>(Result, NAN, 1);
  hipDeviceSynchronize();
  cout << "__hmin_nan(nan, 1) = " << *Result << " (expect nan)";
  check(isnan(*Result));
  hmin_nan<<<1, 1>>>(Result, 1, NAN);
  hipDeviceSynchronize();
  cout << "__hmin_nan(1, nan) = " << *Result << " (expect nan)";
  check(isnan(*Result));
  for (const auto &TestCase : TestCases) {
    hmin_nan<<<1, 1>>>(Result, TestCase.first.first, TestCase.first.second);
    hipDeviceSynchronize();
    checkResult("__hmin_nan", {TestCase.first.first, TestCase.first.second},
                TestCase.second.first, *Result, TestCase.second.second);
  }
}

__global__ void hne(bool *const Result, __hip_bfloat16 Input1,
                    __hip_bfloat16 Input2) {
  *Result = __hne(Input1, Input2);
}

void testHneCases(
    const vector<pair<pair<__hip_bfloat16, __hip_bfloat16>, bool>> &TestCases) {
  bool *Result;
  hipMallocManaged(&Result, sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    hne<<<1, 1>>>(Result, TestCase.first.first, TestCase.first.second);
    hipDeviceSynchronize();
    checkResult("__hne", {TestCase.first.first, TestCase.first.second},
                TestCase.second, *Result);
  }
}

__global__ void hneu(bool *const Result, __hip_bfloat16 Input1,
                     __hip_bfloat16 Input2) {
  *Result = __hneu(Input1, Input2);
}

void testHneuCases(
    const vector<pair<pair<__hip_bfloat16, __hip_bfloat16>, bool>> &TestCases) {
  bool *Result;
  hipMallocManaged(&Result, sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    hneu<<<1, 1>>>(Result, TestCase.first.first, TestCase.first.second);
    hipDeviceSynchronize();
    checkResult("__hneu", {TestCase.first.first, TestCase.first.second},
                TestCase.second, *Result);
  }
}

// Bfloat16 Math Functions

__global__ void _hceil(float *const Result, __hip_bfloat16 Input1) {
  *Result = hceil(Input1);
}

void testHceilCases(const vector<pair<__hip_bfloat16, bf16i_pair>> &TestCases) {
  float *Result;
  hipMallocManaged(&Result, sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    _hceil<<<1, 1>>>(Result, TestCase.first);
    hipDeviceSynchronize();
    checkResult("hceil", {TestCase.first}, TestCase.second.first, *Result,
                TestCase.second.second);
  }
}

__global__ void _hcos(float *const Result, __hip_bfloat16 Input1) {
  *Result = hcos(Input1);
}

void testHcosCases(const vector<pair<__hip_bfloat16, bf16i_pair>> &TestCases) {
  float *Result;
  hipMallocManaged(&Result, sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    _hcos<<<1, 1>>>(Result, TestCase.first);
    hipDeviceSynchronize();
    checkResult("hcos", {TestCase.first}, TestCase.second.first, *Result,
                TestCase.second.second);
  }
}

__global__ void _hexp(float *const Result, __hip_bfloat16 Input1) {
  *Result = hexp(Input1);
}

void testHexpCases(const vector<pair<__hip_bfloat16, bf16i_pair>> &TestCases) {
  float *Result;
  hipMallocManaged(&Result, sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    _hexp<<<1, 1>>>(Result, TestCase.first);
    hipDeviceSynchronize();
    checkResult("hexp", {TestCase.first}, TestCase.second.first, *Result,
                TestCase.second.second);
  }
}

__global__ void _hexp10(float *const Result, __hip_bfloat16 Input1) {
  *Result = hexp10(Input1);
}

void testHexp10Cases(const vector<pair<__hip_bfloat16, bf16i_pair>> &TestCases) {
  float *Result;
  hipMallocManaged(&Result, sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    _hexp10<<<1, 1>>>(Result, TestCase.first);
    hipDeviceSynchronize();
    checkResult("hexp10", {TestCase.first}, TestCase.second.first, *Result,
                TestCase.second.second);
  }
}

__global__ void _hexp2(float *const Result, __hip_bfloat16 Input1) {
  *Result = hexp2(Input1);
}

void testHexp2Cases(const vector<pair<__hip_bfloat16, bf16i_pair>> &TestCases) {
  float *Result;
  hipMallocManaged(&Result, sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    _hexp2<<<1, 1>>>(Result, TestCase.first);
    hipDeviceSynchronize();
    checkResult("hexp2", {TestCase.first}, TestCase.second.first, *Result,
                TestCase.second.second);
  }
}

__global__ void _hfloor(float *const Result, __hip_bfloat16 Input1) {
  *Result = hfloor(Input1);
}

void testHfloorCases(const vector<pair<__hip_bfloat16, bf16i_pair>> &TestCases) {
  float *Result;
  hipMallocManaged(&Result, sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    _hfloor<<<1, 1>>>(Result, TestCase.first);
    hipDeviceSynchronize();
    checkResult("hfloor", {TestCase.first}, TestCase.second.first, *Result,
                TestCase.second.second);
  }
}

__global__ void _hlog(float *const Result, __hip_bfloat16 Input1) {
  *Result = hlog(Input1);
}

void testHlogCases(const vector<pair<__hip_bfloat16, bf16i_pair>> &TestCases) {
  float *Result;
  hipMallocManaged(&Result, sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    _hlog<<<1, 1>>>(Result, TestCase.first);
    hipDeviceSynchronize();
    checkResult("hlog", {TestCase.first}, TestCase.second.first, *Result,
                TestCase.second.second);
  }
}

__global__ void _hlog10(float *const Result, __hip_bfloat16 Input1) {
  *Result = hlog10(Input1);
}

void testHlog10Cases(const vector<pair<__hip_bfloat16, bf16i_pair>> &TestCases) {
  float *Result;
  hipMallocManaged(&Result, sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    _hlog10<<<1, 1>>>(Result, TestCase.first);
    hipDeviceSynchronize();
    checkResult("hlog10", {TestCase.first}, TestCase.second.first, *Result,
                TestCase.second.second);
  }
}

__global__ void _hlog2(float *const Result, __hip_bfloat16 Input1) {
  *Result = hlog2(Input1);
}

void testHlog2Cases(const vector<pair<__hip_bfloat16, bf16i_pair>> &TestCases) {
  float *Result;
  hipMallocManaged(&Result, sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    _hlog2<<<1, 1>>>(Result, TestCase.first);
    hipDeviceSynchronize();
    checkResult("hlog2", {TestCase.first}, TestCase.second.first, *Result,
                TestCase.second.second);
  }
}

__global__ void _hrcp(float *const Result, __hip_bfloat16 Input1) {
  *Result = hrcp(Input1);
}

void testHrcpCases(const vector<pair<__hip_bfloat16, bf16i_pair>> &TestCases) {
  float *Result;
  hipMallocManaged(&Result, sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    _hrcp<<<1, 1>>>(Result, TestCase.first);
    hipDeviceSynchronize();
    checkResult("hrcp", {TestCase.first}, TestCase.second.first, *Result,
                TestCase.second.second);
  }
}

__global__ void _hrint(float *const Result, __hip_bfloat16 Input1) {
  *Result = hrint(Input1);
}

void testHrintCases(const vector<pair<__hip_bfloat16, bf16i_pair>> &TestCases) {
  float *Result;
  hipMallocManaged(&Result, sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    _hrint<<<1, 1>>>(Result, TestCase.first);
    hipDeviceSynchronize();
    checkResult("hrint", {TestCase.first}, TestCase.second.first, *Result,
                TestCase.second.second);
  }
}

__global__ void _hrsqrt(float *const Result, __hip_bfloat16 Input1) {
  *Result = hrsqrt(Input1);
}

void testHrsqrtCases(const vector<pair<__hip_bfloat16, bf16i_pair>> &TestCases) {
  float *Result;
  hipMallocManaged(&Result, sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    _hrsqrt<<<1, 1>>>(Result, TestCase.first);
    hipDeviceSynchronize();
    checkResult("hrsqrt", {TestCase.first}, TestCase.second.first, *Result,
                TestCase.second.second);
  }
}

__global__ void _hsin(float *const Result, __hip_bfloat16 Input1) {
  *Result = hsin(Input1);
}

void testHsinCases(const vector<pair<__hip_bfloat16, bf16i_pair>> &TestCases) {
  float *Result;
  hipMallocManaged(&Result, sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    _hsin<<<1, 1>>>(Result, TestCase.first);
    hipDeviceSynchronize();
    checkResult("hsin", {TestCase.first}, TestCase.second.first, *Result,
                TestCase.second.second);
  }
}

__global__ void _hsqrt(float *const Result, __hip_bfloat16 Input1) {
  *Result = hsqrt(Input1);
}

void testHsqrtCases(const vector<pair<__hip_bfloat16, bf16i_pair>> &TestCases) {
  float *Result;
  hipMallocManaged(&Result, sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    _hsqrt<<<1, 1>>>(Result, TestCase.first);
    hipDeviceSynchronize();
    checkResult("hsqrt", {TestCase.first}, TestCase.second.first, *Result,
                TestCase.second.second);
  }
}

__global__ void _htrunc(float *const Result, __hip_bfloat16 Input1) {
  *Result = htrunc(Input1);
}

void testHtruncCases(const vector<pair<__hip_bfloat16, bf16i_pair>> &TestCases) {
  float *Result;
  hipMallocManaged(&Result, sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    _htrunc<<<1, 1>>>(Result, TestCase.first);
    hipDeviceSynchronize();
    checkResult("htrunc", {TestCase.first}, TestCase.second.first, *Result,
                TestCase.second.second);
  }
}

int main() {
  testHabsCases({
      {{-0.3}, {0.30078125, 16}},
      {{0.3}, {0.30078125, 16}},
      {{0.5}, {0.5, 16}},
      {{0.4}, {0.400390625, 16}},
      {{6}, {6, 15}},
  });
  testHaddCases({
      {{-0.3, -0.4}, {-0.703125, 16}},
      {{0.3, -0.4}, {-0.099609375, 17}},
      {{0.3, 0.4}, {0.703125, 16}},
      {{0.3, 0.8}, {1.1015625, 15}},
      {{3, 4}, {7, 15}},
  });
  testHadd_rnCases({
      {{-0.3, -0.4}, {-0.703125, 16}},
      {{0.3, -0.4}, {-0.099609375, 17}},
      {{0.3, 0.4}, {0.703125, 16}},
      {{0.3, 0.8}, {1.1015625, 15}},
      {{3, 4}, {7, 15}},
  });
  testHadd_satCases({
      {{-0.3, -0.4}, {0, 37}},
      {{0.3, -0.4}, {0, 37}},
      {{0.3, 0.4}, {0.703125, 16}},
      {{0.3, 0.8}, {1, 15}},
      {{3, 4}, {1, 15}},
  });
  testHdivCases({
      {{-0.3, -0.4}, {0.75, 16}},
      {{0.3, -0.4}, {-0.75, 16}},
      {{0.3, 0.4}, {0.75, 16}},
      {{0.3, 0.8}, {0.375, 16}},
      {{3, 4}, {0.75, 16}},
  });
  testHfmaCases({
      {{-0.3, -0.4, -0.2}, {-0.07958984375, 17}},
      {{0.3, -0.4, -0.1}, {-0.220703125, 16}},
      {{0.3, 0.4, 0.1}, {0.220703125, 16}},
      {{0.3, 0.4, 0}, {0.12060546875, 17}},
      {{3, 4, 5}, {17, 14}},
  });
  testHfma_satCases({
      {{-0.3, -0.4, -0.2}, {0, 37}},
      {{0.3, -0.4, -0.1}, {0, 37}},
      {{0.3, 0.4, 0.1}, {0.220703125, 16}},
      {{0.3, 0.4, 0}, {0.12060546875, 17}},
      {{3, 4, 5}, {1, 15}},
  });
  testHmulCases({
      {{-0.3, -0.4}, {0.12060546875, 17}},
      {{0.3, -0.4}, {-0.12060546875, 17}},
      {{0.3, 0.4}, {0.12060546875, 17}},
      {{0.3, 0.8}, {0.2412109375, 16}},
      {{3, 4}, {12, 15}},
  });
  testHmul_rnCases({
      {{-0.3, -0.4}, {0.12060546875, 17}},
      {{0.3, -0.4}, {-0.12060546875, 17}},
      {{0.3, 0.4}, {0.12060546875, 17}},
      {{0.3, 0.8}, {0.2412109375, 16}},
      {{3, 4}, {12, 15}},
  });
  testHmul_satCases({
      {{-0.3, -0.4}, {0.12060546875, 17}},
      {{0.3, -0.4}, {0, 37}},
      {{0.3, 0.4}, {0.12060546875, 17}},
      {{0.3, 0.8}, {0.2412109375, 16}},
      {{3, 4}, {1, 15}},
  });
  testHnegCases({
      {{-0.3}, {0.30078125, 16}},
      {{0.3}, {-0.30078125, 16}},
      {{0.5}, {-0.5, 16}},
      {{0.4}, {-0.400390625, 16}},
      {{6}, {-6, 15}},
  });
  testHsubCases({
      {{-0.3, -0.4}, {0.099609375, 17}},
      {{0.3, -0.4}, {0.703125, 16}},
      {{0.3, 0.4}, {-0.099609375, 17}},
      {{0.3, -0.8}, {1.1015625, 15}},
      {{3, 4}, {-1, 15}},
  });
  testHsub_rnCases({
      {{-0.3, -0.4}, {0.099609375, 17}},
      {{0.3, -0.4}, {0.703125, 16}},
      {{0.3, 0.4}, {-0.099609375, 17}},
      {{0.3, -0.8}, {1.1015625, 15}},
      {{3, 4}, {-1, 15}},
  });
  testHsub_satCases({
      {{-0.3, -0.4}, {0.099609375, 17}},
      {{0.3, -0.4}, {0.703125, 16}},
      {{0.3, 0.4}, {0, 37}},
      {{0.3, -0.8}, {1, 15}},
      {{3, 4}, {0, 37}},
  });
  testHeqCases({
      {{0, -0.4}, false},
      {{0.7, 0.4}, false},
      {{0.7, 0.7}, true},
      {{1, 4}, false},
      {{NAN, 1}, false},
  });
  testHequCases({
      {{0, -0.4}, false},
      {{0.7, 0.4}, false},
      {{0.7, 0.7}, true},
      {{1, 4}, false},
      {{NAN, 1}, true},
  });
  testHgeCases({
      {{0, -0.4}, true},
      {{0.7, 0.4}, true},
      {{0.7, 0.7}, true},
      {{1, 4}, false},
      {{NAN, 1}, false},
  });
  testHgeuCases({
      {{0, -0.4}, true},
      {{0.7, 0.4}, true},
      {{0.7, 0.7}, true},
      {{1, 4}, false},
      {{NAN, 1}, true},
  });
  testHgtCases({
      {{0, -0.4}, true},
      {{0.7, 0.4}, true},
      {{0.7, 0.7}, false},
      {{1, 4}, false},
      {{NAN, 1}, false},
  });
  testHgtuCases({
      {{0, -0.4}, true},
      {{0.7, 0.4}, true},
      {{0.7, 0.7}, false},
      {{1, 4}, false},
      {{NAN, 1}, true},
  });
  testHisinfCases({
      {-0.3, false},
      {0.34, false},
      {0.8, false},
      {INFINITY, true},
      {NAN, false},
  });
  testHisnanCases({
      {-0.3, false},
      {0.34, false},
      {0.8, false},
      {INFINITY, false},
      {NAN, true},
  });
  testHleCases({
      {{0, -0.4}, false},
      {{0.7, 0.4}, false},
      {{0.7, 0.7}, true},
      {{1, 4}, true},
      {{NAN, 1}, false},
  });
  testHleuCases({
      {{0, -0.4}, false},
      {{0.7, 0.4}, false},
      {{0.7, 0.7}, true},
      {{1, 4}, true},
      {{NAN, 1}, true},
  });
  testHltCases({
      {{0, -0.4}, false},
      {{0.7, 0.4}, false},
      {{0.7, 0.7}, false},
      {{1, 4}, true},
      {{NAN, 1}, false},
  });
  testHltuCases({
      {{0, -0.4}, false},
      {{0.7, 0.4}, false},
      {{0.7, 0.7}, false},
      {{1, 4}, true},
      {{NAN, 1}, true},
  });
  testHmaxCases({
      {{0, -0.4}, {0, 37}},
      {{0.7, 0.7}, {0.69921875, 16}},
      {{1, 4}, {4, 15}},
      {{NAN, 1}, {1, 15}},
      {{1, NAN}, {1, 15}},
  });
  testHmax_nanCases({
      {{0, -0.4}, {0, 37}},
      {{0.7, 0.7}, {0.69921875, 16}},
      {{1, 4}, {4, 15}},
  });
  testHminCases({
      {{0, -0.4}, {-0.400390625, 16}},
      {{0.7, 0.7}, {0.69921875, 16}},
      {{1, 4}, {1, 15}},
      {{NAN, 1}, {1, 15}},
      {{1, NAN}, {1, 15}},
  });
  testHmin_nanCases({
      {{0, -0.4}, {-0.400390625, 16}},
      {{0.7, 0.7}, {0.69921875, 16}},
      {{1, 4}, {1, 15}},
  });
  testHneCases({
      {{0, -0.4}, true},
      {{0.7, 0.4}, true},
      {{0.7, 0.7}, false},
      {{1, 4}, true},
      {{NAN, 1}, false},
  });
  testHneuCases({
      {{0, -0.4}, true},
      {{0.7, 0.4}, true},
      {{0.7, 0.7}, false},
      {{1, 4}, true},
      {{NAN, 1}, true},
  });
  testHceilCases({
      {-0.3, {0, 37}},
      {0.34, {1, 15}},
      {0.8, {1, 15}},
      {23, {23, 14}},
      {-12, {-12, 15}},
  });
  testHcosCases({
      {-0.3, {0.96, 2}},
      {0.34, {0.94, 2}},
      {0.8, {0.695, 3}},
      {23, {-0.53, 2}},
      {-12, {0.844, 3}},
  });
  testHexpCases({
      {-0.3, {0.74, 2}},
      {0.34, {1.41, 2}},
      {0.8, {2.23, 2}},
      {10, {22016, -2}},
      {-12, {0.00000614, 8}},
  });
  testHexp10Cases({
      {-0.3, {0.5, 3}},
      {0.34, {2.188, 3}},
      {0.8, {6.31, 2}},
      {4, {9984, -2}},
      {-12, {0.000000000001, 14}},
  });
  testHexp2Cases({
      {-0.3, {0.812, 3}},
      {0.34, {1.2656, 4}},
      {0.8, {1.742, 3}},
      {12, {4096, 0}},
      {-12, {0.0002441, 7}},
  });
  testHfloorCases({
      {-0.3, {-1, 15}},
      {0.34, {0, 37}},
      {0.8, {0, 37}},
      {23, {23, 14}},
      {-12, {-12, 15}},
  });
  testHlogCases({
      {0.3, {-1.2, 2}},
      {0.34, {-1.08, 2}},
      {0.8, {-0.222, 3}},
      {23, {3.14, 2}},
      {12, {2.484, 3}},
  });
  testHlog10Cases({
      {0.3, {-0.52, 2}},
      {0.34, {-0.4688, 4}},
      {0.8, {-0.097, 3}},
      {23, {1.36, 2}},
      {12, {1.08, 2}},
  });
  testHlog2Cases({
      {0.3, {-1.73, 2}},
      {0.34, {-1.6, 1}},
      {0.8, {-0.32, 3}},
      {23, {4.53, 2}},
      {12, {3.58, 2}},
  });
  testHrcpCases({
      {-0.3, {-3.33, 2}},
      {0.34, {2.94, 2}},
      {0.8, {1.25, 2}},
      {23, {0.0435, 4}},
      {-12, {-0.083, 3}},
  });
  testHrintCases({
      {-0.3, {0, 37}},
      {0.34, {0., 37}},
      {0.8, {1, 15}},
      {23, {23, 14}},
      {-12, {-12, 14}},
  });
  testHrsqrtCases({
      {0.3, {1.82, 2}},
      {0.34, {1.72, 2}},
      {0.8, {1.117, 3}},
      {23, {0.209, 3}},
      {12, {0.289, 3}},
  });
  testHsinCases({
      {-0.3, {-0.297, 3}},
      {0.34, {0.334, 3}},
      {0.8, {0.719, 3}},
      {23, {-0.85, 2}},
      {-12, {0.54, 2}},
  });
  testHsqrtCases({
      {0.3, {0.55, 2}},
      {0.34, {0.582, 3}},
      {0.8, {0.895, 3}},
      {23, {4.8, 1}},
      {12, {3.47, 2}},
  });
  testHtruncCases({
      {-0.3, {0, 37}},
      {0.34, {0, 37}},
      {0.8, {0, 37}},
      {23, {23, 14}},
      {-12, {-12, 15}},
  });
  cout << "passed " << passed << "/" << passed + failed << " cases!" << endl;
  if (failed) {
    cout << "failed!" << endl;
  }
  return failed;
}
