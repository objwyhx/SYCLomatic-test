// ====------ cub_device_reduce_sum.cu-------------------- *- CUDA -* ------===//
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
//
// ===----------------------------------------------------------------------===//


#include <iostream>
#include <vector>

#include <hip/hip_runtime.h>
#include <hipcub/hipcub.hpp>

#define DATA_NUM 100


template<typename T>
struct NonZeroOp {
    __host__ __device__ __forceinline__ bool operator()(const T& a) const {
      return (a!=T(0));
    }
};

template<typename T = int>
void init_data(T* data, int num) {
  T host_data[DATA_NUM];
  for(int i = 0; i < num; i++)
    host_data[i] = i;
  hipMemcpy(data, host_data, num * sizeof(T), hipMemcpyHostToDevice);
}
template<typename T = int>
bool verify_data(T* data, T* expect, int num, int step = 1) {
  T host_data[DATA_NUM];
  hipMemcpy(host_data, data, num * sizeof(T), hipMemcpyDeviceToHost);
  for(int i = 0; i < num; i = i + step) {
    if(host_data[i] != expect[i]) {
      return false;
    }
  }
  return true;
}
template<typename T = int>
void print_data(T* data, int num, bool IsHost = false) {
  if(IsHost) {
    for (int i = 0; i < num; i++) {
      std::cout << data[i] << ", ";
      if((i+1)%32 == 0)
        std::cout << std::endl;
    }
    std::cout << std::endl;
    return;
  }
  T host_data[DATA_NUM];
  hipMemcpy(host_data, data, num * sizeof(int), hipMemcpyDeviceToHost);
  for (int i = 0; i < num; i++) {
    std::cout << host_data[i] << ", ";
    if((i+1)%32 == 0)
        std::cout << std::endl;
  }
  std::cout << std::endl;
}

/// hipcub::DeviceReduce::Sum
bool test_device_reduce_sum() {
  int *device_in;
  int *device_out;
  void *temp_storage = NULL;
  size_t temp_storage_size = 0;
  int expect = 4950;
  hipMalloc((void **)&device_in, sizeof(int) * DATA_NUM);
  hipMalloc((void **)&device_out, sizeof(int));
  init_data(device_in, DATA_NUM);
  hipcub::DeviceReduce::Sum(temp_storage, temp_storage_size, device_in,
                         device_out, DATA_NUM);
  hipMalloc((void **)&temp_storage, temp_storage_size);
  hipcub::DeviceReduce::Sum(temp_storage, temp_storage_size, device_in,
                         device_out, DATA_NUM);
  hipDeviceSynchronize();
  if (!verify_data(device_out, &expect, 1)) {
    std::cout << "hipcub::DeviceReduce::Sum verify failed\n";
    std::cout << "expect:\n";
    print_data<int>(&expect, 1, true);
    std::cout << "current result:\n";
    print_data<int>(device_out, 1);
    return false;
  }
  return true;
}

bool test_device_reduce_sum2() {
  int ret = 0;
  float f4[] = {0.1, 0.2, 0.3, 0.4};
  float *d_in;
  int *d_out;
  hipMalloc((void **)&d_in, 4 * sizeof(float));
  hipMalloc((void **)&d_out, sizeof(float));
  hipMemcpy(d_in, f4, sizeof(f4), hipMemcpyHostToDevice);
  hipcub::TransformInputIterator<bool, NonZeroOp<float>, float *> itr(d_in, NonZeroOp<float>());
  void *tmp = nullptr;
  size_t tmp_size = 0;
  hipcub::DeviceReduce::Sum(tmp, tmp_size, itr, d_out, 4);
  hipMalloc((void **)&tmp, tmp_size);
  hipcub::DeviceReduce::Sum(tmp, tmp_size, itr, d_out, 4);
  hipMemcpy(&ret, d_out, sizeof(int), hipMemcpyDeviceToHost);
  return ret == 4;
}

int main() {
  if (test_device_reduce_sum() && test_device_reduce_sum2()) {
    std::cout << "hipcub::DeviceReduce::Sum Pass\n";
    return 0;
  }
  return 1;
}
